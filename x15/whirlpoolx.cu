#include "hip/hip_runtime.h"
/*
 * whirlpool routine (djm)
 * whirlpoolx routine (provos alexis)
 */
extern "C" {
#include "sph/sph_whirlpool.h"
#include "miner.h"
}

#include "cuda_helper.h"

static uint32_t *d_hash[MAX_GPUS];

extern void whirlpoolx_cpu_init(int thr_id, int threads);
extern void whirlpoolx_setBlock_80(void *pdata, const void *ptarget);
extern uint32_t whirlpoolx_cpu_hash(int thr_id, uint32_t threads, uint32_t startNounce);
extern void whirlpoolx_precompute();

// CPU Hash function
extern "C" void whirlxHash(void *state, const void *input)
{
	sph_whirlpool_context ctx_whirlpool;

	unsigned char hash[64];
	unsigned char hash_xored[32];

	memset(hash, 0, sizeof hash);

	sph_whirlpool_init(&ctx_whirlpool);
	sph_whirlpool(&ctx_whirlpool, input, 80);
	sph_whirlpool_close(&ctx_whirlpool, hash);


	for (uint32_t i = 0; i < 32; i++){
	        hash_xored[i] = hash[i] ^ hash[i + 16];
	}
	memcpy(state, hash_xored, 32);
}

static bool init[MAX_GPUS] = { 0 };

extern "C" int scanhash_whirlpoolx(int thr_id, uint32_t *pdata, const uint32_t *ptarget,
	uint32_t max_nonce, unsigned long *hashes_done)
{
	const uint32_t first_nonce = pdata[19];
	uint64_t n = first_nonce;
	uint32_t endiandata[20];
	uint32_t throughput = device_intensity(thr_id, __func__, 1U << 22);
	throughput = min(throughput, max_nonce - first_nonce);

	if (opt_benchmark)
		((uint32_t*)ptarget)[7] = 0x0000ff;

	if (!init[thr_id]) {
		hipSetDevice(device_map[thr_id]);

		hipMalloc(&d_hash[thr_id], 64 * throughput);
		whirlpoolx_cpu_init(thr_id, throughput);

		init[thr_id] = true;
	}

	for (int k=0; k < 20; k++) {
		be32enc(&endiandata[k], pdata[k]);
	}

	whirlpoolx_setBlock_80((void*)endiandata, ptarget);
	whirlpoolx_precompute();
	do {
		uint32_t foundNonce = UINT32_MAX;
		if((n+throughput) >= max_nonce) {
			// Preventing glitch
			throughput = (uint32_t) (max_nonce-n);
		}
		foundNonce = whirlpoolx_cpu_hash(thr_id, throughput, (uint32_t) n);
		if (foundNonce != UINT32_MAX)
		{
			const uint32_t Htarg = ptarget[7];
			uint32_t vhash64[8];
			be32enc(&endiandata[19], foundNonce);
			whirlxHash(vhash64, endiandata);

			if (vhash64[7] <= Htarg && fulltest(vhash64, ptarget)) {
				int res = 1;
				*hashes_done = (unsigned long)(n - first_nonce + throughput);
				pdata[19] = foundNonce;
				return res;
			}
			else if (vhash64[7] > Htarg) {
				applog(LOG_INFO, "GPU #%d: result for %08x is not in range: %x > %x", thr_id, foundNonce, vhash64[7], Htarg);
			}
			else {
				applog(LOG_INFO, "GPU #%d: result for %08x does not validate on CPU!", thr_id, foundNonce);
			}
		}
		n += throughput;

	} while (n < max_nonce && !work_restart[thr_id].restart);
	*hashes_done = (unsigned long)(n - first_nonce);
	return 0;
}
