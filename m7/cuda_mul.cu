#include "hip/hip_runtime.h"
/*
 * tiger-192 djm34
 *
 */

/*
 * tiger-192 kernel implementation.
 *
 * ==========================(LICENSE BEGIN)============================
 *
 * Copyright (c) 2014  djm34
 *
 * Permission is hereby granted, free of charge, to any person obtaining
 * a copy of this software and associated documentation files (the
 * "Software"), to deal in the Software without restriction, including
 * without limitation the rights to use, copy, modify, merge, publish,
 * distribute, sublicense, and/or sell copies of the Software, and to
 * permit persons to whom the Software is furnished to do so, subject to
 * the following conditions:
 *
 * The above copyright notice and this permission notice shall be
 * included in all copies or substantial portions of the Software.
 *
 * THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND,
 * EXPRESS OR IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF
 * MERCHANTABILITY, FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT.
 * IN NO EVENT SHALL THE AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY
 * CLAIM, DAMAGES OR OTHER LIABILITY, WHETHER IN AN ACTION OF CONTRACT,
 * TORT OR OTHERWISE, ARISING FROM, OUT OF OR IN CONNECTION WITH THE
 * SOFTWARE OR THE USE OR OTHER DEALINGS IN THE SOFTWARE.
 *
 * ===========================(LICENSE END)=============================
 *
 * @author   phm <phm@inbox.com>
 */
//#include <stdio.h>
#include <memory.h>

#include "cuda_helper.h"

#define HIWORD _HIWORD
#define LOWORD _LOWORD

#if 0
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, char *file, int line, bool abort=true)
{
   if (code != hipSuccess)
   {
	  fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
	  if (abort) exit(code);
   }
}
#endif

extern hipError_t MyStreamSynchronize(hipStream_t stream, int situation, int thr_id);

__device__ __forceinline__
void bigmul(uint64_t *w, uint64_t* am, uint64_t* bm, int sizea, int sizeb, int thread)
{
	int threads = 256*256*8*2;
	#pragma unroll
	for (int i=0;i<sizea+sizeb;i++) {w[i*threads+thread]=0;}
	#pragma unroll
	for (int i=0;i<sizeb;i++)
	{
		uint64_t c=0;
		uint64_t u=0,v=0;
		#pragma unroll
		for (int j=0;j<sizea;j++) {
			muladd128(u,v,am[j*threads+thread],bm[i*threads+thread],w[(i+j)*threads+thread],c);
			w[(i+j)*threads+thread]=v;
			c=u;
		}
		w[(i+sizea)*threads+thread]=u;
	}
}

__global__
void m7_bigmul1_gpu(int threads, int sizea, int sizeb, uint64_t* am, uint64_t* bm, uint64_t *w)
{
	int thread = (blockDim.x * blockIdx.x + threadIdx.x);
	if (thread < threads)
	{
		#pragma unroll
		for (int i=0;i<sizea+sizeb;i++) {w[i*threads+thread]=0;}
		#pragma unroll
		for (int i=0;i<sizeb;i++) {
			uint64_t c=0;
			uint64_t u=0,v=0;
			#pragma unroll
			for (int j=0;j<sizea;j++) {
				muladd128(u,v,am[j*threads+thread],bm[i*threads+thread],w[(i+j)*threads+thread],c);
				w[(i+j)*threads+thread]=v;
				c=u;
			}
			w[(i+sizea)*threads+thread]=u;
		}
	} // thread
}

__global__
void m7_bigmul_unroll1_gpu(int threads, uint64_t* am, uint64_t* bm, uint64_t *w)
{
	int thread = (blockDim.x * blockIdx.x + threadIdx.x);
	if (thread < threads)
	{
		#pragma unroll 32
		for (int i=0;i<32;i++) {
			w[i*threads + thread]=0;
		}

		#if __CUDA_ARCH__ < 500
		#pragma unroll 32
		#endif
		for (int i=0;i<32;i++)
		{
			uint64_t c=0;
			uint64_t u=0,v=0;
			#pragma unroll 3
			for (int j=0;j<3;j++) {
				muladd128(u,v,am[j*threads+thread],bm[i*threads+thread],w[(i+j)*threads+thread],c);
				w[(i+j)*threads+thread]=v;
				c=u;
			}
			w[(i+3)*threads+thread]=u;
		}
	} // threads
}

__global__
void m7_bigmul_unroll1_gpu_std(int threads, uint64_t* amg, uint64_t* bmg, uint64_t *wg)
{
	int thread = (blockDim.x * blockIdx.x + threadIdx.x);
	if (thread < threads)
	{
		uint64_t * am = amg + 8*thread;
		uint64_t * bm = bmg + 38*thread;
		uint64_t * w  = wg +  38*thread;

		#pragma unroll 32
		for (int i=0;i<32;i++) {
			w[i]=0;
		}

		#if __CUDA_ARCH__ < 500
		#pragma unroll 32
		#endif
		for (int i=0;i<32;i++)
		{
			uint64_t c=0;
			uint64_t u=0,v=0;
			#pragma unroll 3
			for (int j=0;j<3;j++) {
				muladd128(u,v,am[j],bm[i],w[(i+j)],c);
				w[(i+j)]=v;
				c=u;
			}
			w[(i+3)]=u;
		}
	} // threads
}

__global__
void m7_bigmul_unroll2_gpu(int threads, uint64_t* am, uint64_t* bm, uint64_t *w)
{
	int thread = (blockDim.x * blockIdx.x + threadIdx.x);
	if (thread < threads)
	{
		#if __CUDA_ARCH__ < 500
		#pragma unroll
		#endif
		for (int i=0;i<38;i++) {
			w[i*threads+thread]=0;
		}

		#if __CUDA_ARCH__ < 500
		#pragma unroll
		#endif
		for (int i=0;i<35;i++)
		{
			uint64_t c=0;
			uint64_t u=0,v=0;
			#if __CUDA_ARCH__ < 500
			#pragma unroll
			#endif
			for (int j=0;j<3;j++) {
				muladd128(u,v,am[j*threads+thread],bm[i*threads+thread],w[(i+j)*threads+thread],c);
				w[(i+j)*threads+thread]=v;
				c=u;
			}
			w[(i+3)*threads+thread]=u;
		}
	} // thread
}

__global__
void m7_bigmul_unroll2_gpu_std(int threads, uint64_t* amg, uint64_t* bmg, uint64_t *wg)
{
	int thread = (blockDim.x * blockIdx.x + threadIdx.x);

	if (thread < threads)
	{
		uint64_t * am = amg + 8*thread;
		uint64_t * bm = bmg + 38*thread;
		uint64_t * w  = wg +  38*thread;

		#if __CUDA_ARCH__ < 500
		#pragma unroll
		#endif
		for (int i=0;i<38;i++) {
			w[i]=0;
		}

		#if __CUDA_ARCH__ < 500
		#pragma unroll
		#endif
		for (int i=0;i<35;i++)
		{
			uint64_t c=0;
			uint64_t u=0,v=0;
			#if __CUDA_ARCH__ < 500
			#pragma unroll
			#endif
			for (int j=0;j<3;j++) {
				muladd128(u,v,am[j],bm[i],w[(i+j)],c);
				w[(i+j)]=v;
				c=u;
			}
			w[(i+3)]=u;
		}
	} // thread
}

__host__ void m7_bigmul1_cpu(int thr_id, int threads,int len1,int len2,uint64_t* Hash1, uint64_t* Hash2,uint64_t *finalHash,int order)
{
	const int threadsperblock = 256;

	dim3 grid((threads + threadsperblock-1)/threadsperblock);
	dim3 block(threadsperblock);

	size_t shared_size =0;
	m7_bigmul1_gpu<<<grid, block, shared_size>>>(threads,len1,len2,Hash1,Hash2,finalHash);

//	MyStreamSynchronize(NULL, order, thr_id);

//	gpuErrchk(hipDeviceSynchronize());
//	gpuErrchk(hipDeviceSynchronize());
}

__host__ void m7_bigmul_unroll1_cpu(int thr_id, int threads,uint64_t* Hash1, uint64_t* Hash2,uint64_t *finalHash,int order)
{
	const int threadsperblock = 256;

	dim3 grid((threads + threadsperblock-1)/threadsperblock);
	dim3 block(threadsperblock);

	size_t shared_size =0;
	m7_bigmul_unroll1_gpu<<<grid, block, shared_size>>>(threads,Hash1,Hash2,finalHash);
}

__host__ void m7_bigmul_unroll2_cpu(int thr_id, int threads,uint64_t* Hash1, uint64_t* Hash2,uint64_t *finalHash,int order)
{
	const int threadsperblock = 256;

	dim3 grid((threads + threadsperblock-1)/threadsperblock);
	dim3 block(threadsperblock);

	size_t shared_size =0;
	m7_bigmul_unroll2_gpu<<<grid, block, shared_size>>>(threads,Hash1,Hash2,finalHash);
}

__host__ void m7_bigmul_init(int thr_id, int threads)
{
	// why I am here ?
}
