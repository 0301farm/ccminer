#include "hip/hip_runtime.h"
extern "C" {
#include "sph/sph_skein.h"
}

#include "miner.h"
#include "cuda_helper.h"

static uint32_t *d_hash[MAX_GPUS];

extern void skein512_cpu_setBlock_80(void *pdata);
extern void skein512_cpu_hash_80(int thr_id, uint32_t threads, uint32_t startNounce, uint32_t *d_hash, int order);

extern void quark_skein512_cpu_hash_64(int thr_id, uint32_t threads, uint32_t startNounce, uint32_t *d_nonceVector, uint32_t *d_hash, int order);

extern "C" void skein2hash(void *output, const void *input)
{
	sph_skein512_context ctx_skein;

	uint32_t hash[16];

	sph_skein512_init(&ctx_skein);
	sph_skein512(&ctx_skein, input, 80);
	sph_skein512_close(&ctx_skein, hash);
	//applog_hash((uchar*)hash);
	//applog_hash((uchar*)&hash[8]);
	sph_skein512_init(&ctx_skein);
	sph_skein512(&ctx_skein, hash, 64);
	sph_skein512_close(&ctx_skein, hash);

	memcpy(output, hash, 32);
}

#ifdef _DEBUG
#define TRACE(algo) { \
	if (max_nonce == 1 && pdata[19] <= 1) { \
		uint32_t* debugbuf = NULL; \
		hipHostMalloc(&debugbuf, 16*sizeof(uint32_t)); \
		hipMemcpy(debugbuf, d_hash[thr_id], 16*sizeof(uint32_t), hipMemcpyDeviceToHost); \
		printf("SK2 %s %08x %08x %08x %08x...\n", algo, \
			swab32(debugbuf[0]), swab32(debugbuf[1]), swab32(debugbuf[2]), swab32(debugbuf[3])); \
		hipFree(debugbuf); \
		} \
}
#else
#define TRACE(algo) {}
#endif

static bool init[MAX_GPUS] = { 0 };

extern "C" int scanhash_skein2(int thr_id, uint32_t *pdata,
    const uint32_t *ptarget, uint32_t max_nonce,
    unsigned long *hashes_done)
{
	const uint32_t first_nonce = pdata[19];

	uint32_t throughput =  device_intensity(thr_id, __func__, 1 << 19); // 256*256*8
	throughput = min(throughput,  (max_nonce - first_nonce));

	if (opt_benchmark)
		((uint32_t*)ptarget)[7] = 0x0FFF;

	if (!init[thr_id])
	{
		hipDeviceReset();
		hipSetDevice(device_map[thr_id]);

		CUDA_SAFE_CALL(hipMalloc(&d_hash[thr_id], 64 * throughput));

		cuda_check_cpu_init(thr_id, throughput);
		init[thr_id] = true;
	}

	uint32_t endiandata[20];
	for (int k=0; k < 20; k++)
		be32enc(&endiandata[k], pdata[k]);

	skein512_cpu_setBlock_80((void*)endiandata);
	cuda_check_cpu_setTarget(ptarget);

	do {
		int order = 0;

		// Hash with CUDA
		skein512_cpu_hash_80(thr_id, throughput, pdata[19], d_hash[thr_id], order++);
		TRACE("80:");
		quark_skein512_cpu_hash_64(thr_id, throughput, pdata[19], NULL, d_hash[thr_id], order++);
		TRACE("64:");

		uint32_t foundNonce = cuda_check_hash(thr_id, throughput, pdata[19], d_hash[thr_id]);
		if (foundNonce != UINT32_MAX)
		{
			uint32_t vhash64[8];

			endiandata[19] = foundNonce;
			skein2hash(vhash64, endiandata);

			#define Htarg ptarget[7]
			if (vhash64[7] <= Htarg && fulltest(vhash64, ptarget)) {
				int res = 1;
				uint32_t secNonce = cuda_check_hash_suppl(thr_id, throughput, pdata[19], d_hash[thr_id], 1);
				*hashes_done = pdata[19] - first_nonce + throughput;
				if (secNonce != 0) {
					if (!opt_quiet)
						applog(LOG_BLUE, "GPU #%d: found second nonce %08x !", device_map[thr_id], swab32(secNonce));
					pdata[21] = swab32(secNonce);
					res++;
				}
				pdata[19] = swab32(foundNonce);
				return res;
			}
			else {
				applog(LOG_INFO, "GPU #%d: result for nonce $%08X does not validate on CPU!", device_map[thr_id], foundNonce);
			}
		}

		pdata[19] += throughput;

	} while (pdata[19] < max_nonce && !work_restart[thr_id].restart);

	*hashes_done = pdata[19] - first_nonce + 1;
	return 0;
}
