#include "hip/hip_runtime.h"
/*
 * sha256 CUDA implementation.
 */
#include <stdio.h>
#include <stdint.h>
#include <memory.h>

#include <cuda_helper.h>
#include <miner.h>

__constant__ static uint32_t __align__(8) c_midstate112[8];
__constant__ static uint32_t __align__(8) c_dataEnd112[12];

const __constant__  uint32_t __align__(8) c_H256[8] = {
	0x6A09E667U, 0xBB67AE85U, 0x3C6EF372U, 0xA54FF53AU,
	0x510E527FU, 0x9B05688CU, 0x1F83D9ABU, 0x5BE0CD19U
};
__constant__ static uint32_t __align__(8) c_K[64];

static __thread uint32_t* d_resNonces;
__constant__ static uint32_t __align__(8) c_target[2];
__device__ uint64_t d_target[1];

// ------------------------------------------------------------------------------------------------

static const uint32_t cpu_H256[8] = {
	0x6A09E667U, 0xBB67AE85U, 0x3C6EF372U, 0xA54FF53AU,
	0x510E527FU, 0x9B05688CU, 0x1F83D9ABU, 0x5BE0CD19U
};

static const uint32_t cpu_K[64] = {
	0x428A2F98, 0x71374491, 0xB5C0FBCF, 0xE9B5DBA5, 0x3956C25B, 0x59F111F1, 0x923F82A4, 0xAB1C5ED5,
	0xD807AA98, 0x12835B01, 0x243185BE, 0x550C7DC3, 0x72BE5D74, 0x80DEB1FE, 0x9BDC06A7, 0xC19BF174,
	0xE49B69C1, 0xEFBE4786, 0x0FC19DC6, 0x240CA1CC, 0x2DE92C6F, 0x4A7484AA, 0x5CB0A9DC, 0x76F988DA,
	0x983E5152, 0xA831C66D, 0xB00327C8, 0xBF597FC7, 0xC6E00BF3, 0xD5A79147, 0x06CA6351, 0x14292967,
	0x27B70A85, 0x2E1B2138, 0x4D2C6DFC, 0x53380D13, 0x650A7354, 0x766A0ABB, 0x81C2C92E, 0x92722C85,
	0xA2BFE8A1, 0xA81A664B, 0xC24B8B70, 0xC76C51A3, 0xD192E819, 0xD6990624, 0xF40E3585, 0x106AA070,
	0x19A4C116, 0x1E376C08, 0x2748774C, 0x34B0BCB5, 0x391C0CB3, 0x4ED8AA4A, 0x5B9CCA4F, 0x682E6FF3,
	0x748F82EE, 0x78A5636F, 0x84C87814, 0x8CC70208, 0x90BEFFFA, 0xA4506CEB, 0xBEF9A3F7, 0xC67178F2
};

#define ROTR ROTR32

__host__
static void sha256_step1_host(uint32_t a, uint32_t b, uint32_t c, uint32_t &d,
	uint32_t e, uint32_t f, uint32_t g, uint32_t &h,
	uint32_t in, const uint32_t Kshared)
{
	uint32_t t1,t2;
	uint32_t vxandx = (((f) ^ (g)) & (e)) ^ (g); // xandx(e, f, g);
	uint32_t bsg21 = ROTR(e, 6) ^ ROTR(e, 11) ^ ROTR(e, 25); // bsg2_1(e);
	uint32_t bsg20 = ROTR(a, 2) ^ ROTR(a, 13) ^ ROTR(a, 22); //bsg2_0(a);
	uint32_t andorv = ((b) & (c)) | (((b) | (c)) & (a)); //andor32(a,b,c);

	t1 = h + bsg21 + vxandx + Kshared + in;
	t2 = bsg20 + andorv;
	d = d + t1;
	h = t1 + t2;
}

__host__
static void sha256_step2_host(uint32_t a, uint32_t b, uint32_t c, uint32_t &d,
	uint32_t e, uint32_t f, uint32_t g, uint32_t &h,
	uint32_t* in, uint32_t pc, const uint32_t Kshared)
{
	uint32_t t1,t2;

	int pcidx1 = (pc-2)  & 0xF;
	int pcidx2 = (pc-7)  & 0xF;
	int pcidx3 = (pc-15) & 0xF;

	uint32_t inx0 = in[pc];
	uint32_t inx1 = in[pcidx1];
	uint32_t inx2 = in[pcidx2];
	uint32_t inx3 = in[pcidx3];

	uint32_t ssg21 = ROTR(inx1, 17) ^ ROTR(inx1, 19) ^ SPH_T32((inx1) >> 10); //ssg2_1(inx1);
	uint32_t ssg20 = ROTR(inx3, 7) ^ ROTR(inx3, 18) ^ SPH_T32((inx3) >> 3); //ssg2_0(inx3);
	uint32_t vxandx = (((f) ^ (g)) & (e)) ^ (g); // xandx(e, f, g);
	uint32_t bsg21 = ROTR(e, 6) ^ ROTR(e, 11) ^ ROTR(e, 25); // bsg2_1(e);
	uint32_t bsg20 = ROTR(a, 2) ^ ROTR(a, 13) ^ ROTR(a, 22); //bsg2_0(a);
	uint32_t andorv = ((b) & (c)) | (((b) | (c)) & (a)); //andor32(a,b,c);

	in[pc] = ssg21 + inx2 + ssg20 + inx0;

	t1 = h + bsg21 + vxandx + Kshared + in[pc];
	t2 = bsg20 + andorv;
	d =  d + t1;
	h = t1 + t2;
}

__host__
static void sha256_round_body_host(uint32_t* in, uint32_t* state, const uint32_t* Kshared)
{
	uint32_t a = state[0];
	uint32_t b = state[1];
	uint32_t c = state[2];
	uint32_t d = state[3];
	uint32_t e = state[4];
	uint32_t f = state[5];
	uint32_t g = state[6];
	uint32_t h = state[7];

	sha256_step1_host(a,b,c,d,e,f,g,h,in[0], Kshared[0]);
	sha256_step1_host(h,a,b,c,d,e,f,g,in[1], Kshared[1]);
	sha256_step1_host(g,h,a,b,c,d,e,f,in[2], Kshared[2]);
	sha256_step1_host(f,g,h,a,b,c,d,e,in[3], Kshared[3]);
	sha256_step1_host(e,f,g,h,a,b,c,d,in[4], Kshared[4]);
	sha256_step1_host(d,e,f,g,h,a,b,c,in[5], Kshared[5]);
	sha256_step1_host(c,d,e,f,g,h,a,b,in[6], Kshared[6]);
	sha256_step1_host(b,c,d,e,f,g,h,a,in[7], Kshared[7]);
	sha256_step1_host(a,b,c,d,e,f,g,h,in[8], Kshared[8]);
	sha256_step1_host(h,a,b,c,d,e,f,g,in[9], Kshared[9]);
	sha256_step1_host(g,h,a,b,c,d,e,f,in[10],Kshared[10]);
	sha256_step1_host(f,g,h,a,b,c,d,e,in[11],Kshared[11]);
	sha256_step1_host(e,f,g,h,a,b,c,d,in[12],Kshared[12]);
	sha256_step1_host(d,e,f,g,h,a,b,c,in[13],Kshared[13]);
	sha256_step1_host(c,d,e,f,g,h,a,b,in[14],Kshared[14]);
	sha256_step1_host(b,c,d,e,f,g,h,a,in[15],Kshared[15]);

	for (int i=0; i<3; i++)
	{
		sha256_step2_host(a,b,c,d,e,f,g,h,in,0, Kshared[16+16*i]);
		sha256_step2_host(h,a,b,c,d,e,f,g,in,1, Kshared[17+16*i]);
		sha256_step2_host(g,h,a,b,c,d,e,f,in,2, Kshared[18+16*i]);
		sha256_step2_host(f,g,h,a,b,c,d,e,in,3, Kshared[19+16*i]);
		sha256_step2_host(e,f,g,h,a,b,c,d,in,4, Kshared[20+16*i]);
		sha256_step2_host(d,e,f,g,h,a,b,c,in,5, Kshared[21+16*i]);
		sha256_step2_host(c,d,e,f,g,h,a,b,in,6, Kshared[22+16*i]);
		sha256_step2_host(b,c,d,e,f,g,h,a,in,7, Kshared[23+16*i]);
		sha256_step2_host(a,b,c,d,e,f,g,h,in,8, Kshared[24+16*i]);
		sha256_step2_host(h,a,b,c,d,e,f,g,in,9, Kshared[25+16*i]);
		sha256_step2_host(g,h,a,b,c,d,e,f,in,10,Kshared[26+16*i]);
		sha256_step2_host(f,g,h,a,b,c,d,e,in,11,Kshared[27+16*i]);
		sha256_step2_host(e,f,g,h,a,b,c,d,in,12,Kshared[28+16*i]);
		sha256_step2_host(d,e,f,g,h,a,b,c,in,13,Kshared[29+16*i]);
		sha256_step2_host(c,d,e,f,g,h,a,b,in,14,Kshared[30+16*i]);
		sha256_step2_host(b,c,d,e,f,g,h,a,in,15,Kshared[31+16*i]);
	}

	state[0] += a;
	state[1] += b;
	state[2] += c;
	state[3] += d;
	state[4] += e;
	state[5] += f;
	state[6] += g;
	state[7] += h;
}

__device__ __forceinline__
uint32_t xor3b(const uint32_t a, const uint32_t b, const uint32_t c) {
	uint32_t result;
#if __CUDA_ARCH__ >= 500 && CUDA_VERSION >= 7050
	asm ("lop3.b32 %0, %1, %2, %3, 0x96; // xor3b"  //0x96 = 0xF0 ^ 0xCC ^ 0xAA
		: "=r"(result) : "r"(a), "r"(b),"r"(c));
#else
	result = a^b^c;
#endif
	return result;
}

__device__ __forceinline__ uint32_t bsg2_0(const uint32_t x)
{
	uint32_t r1 = ROTR32(x,2);
	uint32_t r2 = ROTR32(x,13);
	uint32_t r3 = ROTR32(x,22);
	return xor3b(r1,r2,r3);
}

__device__ __forceinline__ uint32_t bsg2_1(const uint32_t x)
{
	uint32_t r1 = ROTR32(x,6);
	uint32_t r2 = ROTR32(x,11);
	uint32_t r3 = ROTR32(x,25);
	return xor3b(r1,r2,r3);
}

__device__ __forceinline__ uint32_t ssg2_0(const uint32_t x)
{
	uint64_t r1 = ROTR32(x,7);
	uint64_t r2 = ROTR32(x,18);
	uint64_t r3 = shr_t32(x,3);
	return xor3b(r1,r2,r3);
}

__device__ __forceinline__ uint32_t ssg2_1(const uint32_t x)
{
	uint64_t r1 = ROTR32(x,17);
	uint64_t r2 = ROTR32(x,19);
	uint64_t r3 = shr_t32(x,10);
	return xor3b(r1,r2,r3);
}

__device__ __forceinline__ uint32_t andor32(const uint32_t a, const uint32_t b, const uint32_t c)
{
	uint32_t result;
	asm("{\n\t"
		".reg .u32 m,n,o;\n\t"
		"and.b32 m,  %1, %2;\n\t"
		" or.b32 n,  %1, %2;\n\t"
		"and.b32 o,   n, %3;\n\t"
		" or.b32 %0,  m, o ;\n\t"
		"}\n\t" : "=r"(result) : "r"(a), "r"(b), "r"(c)
	);
	return result;
}

__device__ __forceinline__ uint2 vectorizeswap(uint64_t v) {
	uint2 result;
	asm("mov.b64 {%0,%1},%2; \n\t"
		: "=r"(result.y), "=r"(result.x) : "l"(v));
	return result;
}

__device__
static void sha2_step1(uint32_t a, uint32_t b, uint32_t c, uint32_t &d, uint32_t e, uint32_t f, uint32_t g, uint32_t &h,
	uint32_t in, const uint32_t Kshared)
{
	uint32_t t1,t2;
	uint32_t vxandx = xandx(e, f, g);
	uint32_t bsg21 = bsg2_1(e);
	uint32_t bsg20 = bsg2_0(a);
	uint32_t andorv = andor32(a,b,c);

	t1 = h + bsg21 + vxandx + Kshared + in;
	t2 = bsg20 + andorv;
	d = d + t1;
	h = t1 + t2;
}

__device__
static void sha2_step2(uint32_t a, uint32_t b, uint32_t c, uint32_t &d, uint32_t e, uint32_t f, uint32_t g, uint32_t &h,
	uint32_t* in, uint32_t pc, const uint32_t Kshared)
{
	uint32_t t1,t2;

	int pcidx1 = (pc-2) & 0xF;
	int pcidx2 = (pc-7) & 0xF;
	int pcidx3 = (pc-15) & 0xF;

	uint32_t inx0 = in[pc];
	uint32_t inx1 = in[pcidx1];
	uint32_t inx2 = in[pcidx2];
	uint32_t inx3 = in[pcidx3];

	uint32_t ssg21 = ssg2_1(inx1);
	uint32_t ssg20 = ssg2_0(inx3);
	uint32_t vxandx = xandx(e, f, g);
	uint32_t bsg21 = bsg2_1(e);
	uint32_t bsg20 = bsg2_0(a);
	uint32_t andorv = andor32(a,b,c);

	in[pc] = ssg21 + inx2 + ssg20 + inx0;

	t1 = h + bsg21 + vxandx + Kshared + in[pc];
	t2 = bsg20 + andorv;
	d =  d + t1;
	h = t1 + t2;
}

__device__
static void sha256_round_body(uint32_t* in, uint32_t* state, uint32_t* const Kshared)
{
	uint32_t a = state[0];
	uint32_t b = state[1];
	uint32_t c = state[2];
	uint32_t d = state[3];
	uint32_t e = state[4];
	uint32_t f = state[5];
	uint32_t g = state[6];
	uint32_t h = state[7];

	sha2_step1(a,b,c,d,e,f,g,h,in[0], Kshared[0]);
	sha2_step1(h,a,b,c,d,e,f,g,in[1], Kshared[1]);
	sha2_step1(g,h,a,b,c,d,e,f,in[2], Kshared[2]);
	sha2_step1(f,g,h,a,b,c,d,e,in[3], Kshared[3]);
	sha2_step1(e,f,g,h,a,b,c,d,in[4], Kshared[4]);
	sha2_step1(d,e,f,g,h,a,b,c,in[5], Kshared[5]);
	sha2_step1(c,d,e,f,g,h,a,b,in[6], Kshared[6]);
	sha2_step1(b,c,d,e,f,g,h,a,in[7], Kshared[7]);
	sha2_step1(a,b,c,d,e,f,g,h,in[8], Kshared[8]);
	sha2_step1(h,a,b,c,d,e,f,g,in[9], Kshared[9]);
	sha2_step1(g,h,a,b,c,d,e,f,in[10],Kshared[10]);
	sha2_step1(f,g,h,a,b,c,d,e,in[11],Kshared[11]);
	sha2_step1(e,f,g,h,a,b,c,d,in[12],Kshared[12]);
	sha2_step1(d,e,f,g,h,a,b,c,in[13],Kshared[13]);
	sha2_step1(c,d,e,f,g,h,a,b,in[14],Kshared[14]);
	sha2_step1(b,c,d,e,f,g,h,a,in[15],Kshared[15]);

	#pragma unroll
	for (int i=0; i<3; i++)
	{
		sha2_step2(a,b,c,d,e,f,g,h,in,0, Kshared[16+16*i]);
		sha2_step2(h,a,b,c,d,e,f,g,in,1, Kshared[17+16*i]);
		sha2_step2(g,h,a,b,c,d,e,f,in,2, Kshared[18+16*i]);
		sha2_step2(f,g,h,a,b,c,d,e,in,3, Kshared[19+16*i]);
		sha2_step2(e,f,g,h,a,b,c,d,in,4, Kshared[20+16*i]);
		sha2_step2(d,e,f,g,h,a,b,c,in,5, Kshared[21+16*i]);
		sha2_step2(c,d,e,f,g,h,a,b,in,6, Kshared[22+16*i]);
		sha2_step2(b,c,d,e,f,g,h,a,in,7, Kshared[23+16*i]);
		sha2_step2(a,b,c,d,e,f,g,h,in,8, Kshared[24+16*i]);
		sha2_step2(h,a,b,c,d,e,f,g,in,9, Kshared[25+16*i]);
		sha2_step2(g,h,a,b,c,d,e,f,in,10,Kshared[26+16*i]);
		sha2_step2(f,g,h,a,b,c,d,e,in,11,Kshared[27+16*i]);
		sha2_step2(e,f,g,h,a,b,c,d,in,12,Kshared[28+16*i]);
		sha2_step2(d,e,f,g,h,a,b,c,in,13,Kshared[29+16*i]);
		sha2_step2(c,d,e,f,g,h,a,b,in,14,Kshared[30+16*i]);
		sha2_step2(b,c,d,e,f,g,h,a,in,15,Kshared[31+16*i]);
	}

	state[0] += a;
	state[1] += b;
	state[2] += c;
	state[3] += d;
	state[4] += e;
	state[5] += f;
	state[6] += g;
	state[7] += h;
}

__device__
uint64_t cuda_swab32ll(uint64_t x) {
	return MAKE_ULONGLONG(cuda_swab32(_LODWORD(x)), cuda_swab32(_HIDWORD(x)));
}

__global__
/*__launch_bounds__(256,3)*/
void lbry_sha256_gpu_hash_112(const uint32_t threads, const uint32_t startNonce, const bool swabNonce, uint64_t *outputHash)
{
	const uint32_t thread = (blockDim.x * blockIdx.x + threadIdx.x);
	if (thread < threads)
	{
		const uint32_t nonce = startNonce + thread;

		uint32_t dat[16];
		#pragma unroll
		for (int i=0;i<11;i++) dat[i] = c_dataEnd112[i]; // pre "swabed"
		dat[11] = swabNonce ? cuda_swab32(nonce) : nonce;
		dat[12] = 0x80000000;
		dat[13] = 0;
		dat[14] = 0;
		dat[15] = 0x380;

		uint32_t __align__(8) buf[8];
		#pragma unroll
		for (int i=0;i<8;i++) buf[i] = c_midstate112[i];

		sha256_round_body(dat, buf, c_K);

		// output
		uint2* output = (uint2*) (&outputHash[thread * 8U]);
		#pragma unroll
		for (int i=0;i<4;i++) {
			//output[i] = vectorize(cuda_swab32ll(((uint64_t*)buf)[i]));
			output[i] = vectorize(((uint64_t*)buf)[i]); // out without swap, new sha256 after
		}
	}
}

__global__
/*__launch_bounds__(256,3)*/
void lbry_sha256_gpu_hash_32(uint32_t threads, uint64_t *Hash512)
{
	const uint32_t thread = (blockDim.x * blockIdx.x + threadIdx.x);
	if (thread < threads)
	{
		uint32_t __align__(8) buf[8]; // align for vectorize
		#pragma unroll
		for (int i=0; i<8; i++) buf[i] = c_H256[i];

		uint32_t* input = (uint32_t*) (&Hash512[thread * 8U]);

		uint32_t dat[16];
		#pragma unroll
		//for (int i=0;i<8;i++) dat[i] = cuda_swab32(input[i]);
		for (int i=0; i<8; i++) dat[i] = input[i];
		dat[8] = 0x80000000;
		#pragma unroll
		for (int i=9; i<15; i++) dat[i] = 0;
		dat[15] = 0x100;

		sha256_round_body(dat, buf, c_K);

		// output
		uint2* output = (uint2*) input;
		#pragma unroll
		for (int i=0;i<4;i++) {
			//output[i] = vectorize(cuda_swab32ll(((uint64_t*)buf)[i]));
			output[i] = vectorizeswap(((uint64_t*)buf)[i]);
		}
#ifdef PAD_ZEROS
		#pragma unroll
		for (int i=4; i<8; i++) output[i] = vectorize(0);
#endif
	}
}

__global__
/*__launch_bounds__(256,3)*/
void lbry_sha256d_gpu_hash_112(const uint32_t threads, const uint32_t startNonce, const bool swabNonce, uint64_t *outputHash)
{
	const uint32_t thread = (blockDim.x * blockIdx.x + threadIdx.x);
	extern __shared__ uint32_t s_K[];
	//s_K[thread & 63] = c_K[thread & 63];
	if (threadIdx.x < 64U) s_K[threadIdx.x] = c_K[threadIdx.x];
	if (thread < threads)
	{
		const uint32_t nonce = startNonce + thread;

		uint32_t dat[16];
		#pragma unroll
		for (int i=0; i<11; i++) dat[i] = c_dataEnd112[i];
		dat[11] = swabNonce ? cuda_swab32(nonce) : nonce;
		dat[12] = 0x80000000;
		dat[13] = 0;
		dat[14] = 0;
		dat[15] = 0x380;

		uint32_t __align__(8) buf[8];
		#pragma unroll
		for (int i=0;i<8;i++) buf[i] = c_midstate112[i];

		sha256_round_body(dat, buf, s_K);

		// second sha256

		#pragma unroll
		for (int i=0; i<8; i++) dat[i] = buf[i];
		dat[8] = 0x80000000;
		#pragma unroll
		for (int i=9; i<15; i++) dat[i] = 0;
		dat[15] = 0x100;

		#pragma unroll
		for (int i=0; i<8; i++) buf[i] = c_H256[i];

		sha256_round_body(dat, buf, s_K);

		// output
		uint2* output = (uint2*) (&outputHash[thread * 8U]);
		#pragma unroll
		for (int i=0;i<4;i++) {
		//	//output[i] = vectorize(cuda_swab32ll(((uint64_t*)buf)[i]));
			output[i] = vectorizeswap(((uint64_t*)buf)[i]);
		}
	}
}

__global__
/*__launch_bounds__(256,3)*/
void lbry_sha256_gpu_hash_20x2(uint32_t threads, uint64_t *Hash512)
{
	const uint32_t thread = (blockDim.x * blockIdx.x + threadIdx.x);
	if (thread < threads)
	{
		uint32_t __align__(8) buf[8]; // align for vectorize
		#pragma unroll
		for (int i=0;i<8;i++) buf[i] = c_H256[i];

		uint32_t* input = (uint32_t*) (&Hash512[thread * 8U]);

		uint32_t dat[16];
		#pragma unroll
		for (int i=0;i<5;i++) dat[i] = cuda_swab32(input[i]);
		#pragma unroll
		for (int i=0;i<5;i++) dat[i+5] = cuda_swab32(input[i+8]);
		dat[10] = 0x80000000;
		#pragma unroll
		for (int i=11;i<15;i++) dat[i] = 0;
		dat[15] = 0x140;

		sha256_round_body(dat, buf, c_K);

		// output
		uint2* output = (uint2*) input;
		#pragma unroll
		for (int i=0;i<4;i++) {
			//output[i] = vectorize(cuda_swab32ll(((uint64_t*)buf)[i]));
			output[i] = vectorize(((uint64_t*)buf)[i]);
		}
#ifdef PAD_ZEROS
		#pragma unroll
		for (int i=4; i<8; i++) output[i] = vectorize(0);
#endif
	}
}

__global__
/*__launch_bounds__(256,3)*/
void lbry_sha256d_gpu_hash_20x2(uint32_t threads, uint64_t *Hash512)
{
	const uint32_t thread = (blockDim.x * blockIdx.x + threadIdx.x);
	extern __shared__ uint32_t s_K[];
	if (threadIdx.x < 64U) s_K[threadIdx.x] = c_K[threadIdx.x];
	if (thread < threads)
	{
		uint32_t __align__(8) buf[8]; // align for vectorize
		#pragma unroll
		for (int i=0;i<8;i++) buf[i] = c_H256[i];

		uint32_t* input = (uint32_t*) (&Hash512[thread * 8U]);

		uint32_t dat[16];
		#pragma unroll
		for (int i=0; i<5; i++) dat[i] = cuda_swab32(input[i]);
		#pragma unroll
		for (int i=0; i<5; i++) dat[i+5] = cuda_swab32(input[i+8]);
		dat[10] = 0x80000000;
		#pragma unroll
		for (int i=11;i<15;i++) dat[i] = 0;
		dat[15] = 0x140;

		sha256_round_body(dat, buf, s_K);

		// second sha256

		#pragma unroll
		for (int i=0; i<8; i++) dat[i] = buf[i];
		dat[8] = 0x80000000;
		#pragma unroll
		for (int i=9; i<15; i++) dat[i] = 0;
		dat[15] = 0x100;

		#pragma unroll
		for (int i=0;i<8;i++) buf[i] = c_H256[i];

		sha256_round_body(dat, buf, s_K);

		// output
		uint2* output = (uint2*) input;

#ifdef FULL_HASH
		#pragma unroll
		for (int i=0;i<4;i++) {
			output[i] = vectorize(cuda_swab32ll(((uint64_t*)buf)[i]));
			//output[i] = vectorize(((uint64_t*)buf)[i]);
		}
#	ifdef PAD_ZEROS
		#pragma unroll
		for (int i=4; i<8; i++) output[i] = vectorize(0);
#	endif

#else
		//input[6] = cuda_swab32(buf[6]);
		//input[7] = cuda_swab32(buf[7]);
		output[3] = vectorize(cuda_swab32ll(((uint64_t*)buf)[3]));
#endif
	}
}

__host__
void lbry_sha256_init(int thr_id)
{
	//hipMemcpyToSymbol(HIP_SYMBOL(c_H256), cpu_H256, sizeof(cpu_H256), 0, hipMemcpyHostToDevice);
	hipMemcpyToSymbol(HIP_SYMBOL(c_K), cpu_K, sizeof(cpu_K), 0, hipMemcpyHostToDevice);
	CUDA_SAFE_CALL(hipMalloc(&d_resNonces, 4*sizeof(uint32_t)));
}

__host__
void lbry_sha256_free(int thr_id)
{
	hipFree(d_resNonces);
}

__host__
void lbry_sha256_setBlock_112(uint32_t *pdata, uint32_t *ptarget)
{
	uint32_t in[16], buf[8], end[11];
	for (int i=0;i<16;i++) in[i] = cuda_swab32(pdata[i]);
	for (int i=0; i<8;i++) buf[i] = cpu_H256[i];
	for (int i=0;i<11;i++) end[i] = cuda_swab32(pdata[16+i]);
	sha256_round_body_host(in, buf, cpu_K);

	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(c_midstate112), buf, 32, 0, hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(c_dataEnd112),  end, sizeof(end), 0, hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(c_target), &ptarget[6], sizeof(uint64_t), 0, hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(d_target), &ptarget[6], sizeof(uint64_t), 0, hipMemcpyHostToDevice));
}

__host__
void lbry_sha256_hash_112(int thr_id, uint32_t threads, uint32_t startNonce, uint32_t *d_outputHash, bool swabNonce, hipStream_t stream)
{
	const int threadsperblock = 256;

	dim3 grid(threads/threadsperblock);
	dim3 block(threadsperblock);

	lbry_sha256_gpu_hash_112 <<<grid, block, 0, stream>>> (threads, startNonce, swabNonce, (uint64_t*) d_outputHash);
	hipGetLastError();
}

__host__
void lbry_sha256_hash_32(int thr_id, uint32_t threads, uint32_t *d_Hash, hipStream_t stream)
{
	const int threadsperblock = 256;

	dim3 grid(threads/threadsperblock);
	dim3 block(threadsperblock);

	lbry_sha256_gpu_hash_32 <<<grid, block, 0, stream>>> (threads, (uint64_t*) d_Hash);
}

__host__
void lbry_sha256d_hash_112(int thr_id, uint32_t threads, uint32_t startNonce, uint32_t *d_outputHash, bool swabNonce, hipStream_t stream)
{
	const int threadsperblock = 256;

	dim3 grid(threads/threadsperblock);
	dim3 block(threadsperblock);

	lbry_sha256d_gpu_hash_112 <<<grid, block, 64*4, stream>>> (threads, startNonce, swabNonce, (uint64_t*) d_outputHash);
}

__host__
void lbry_sha256_hash_20x2(int thr_id, uint32_t threads, uint32_t *d_Hash, hipStream_t stream)
{
	const int threadsperblock = 256;

	dim3 grid(threads/threadsperblock);
	dim3 block(threadsperblock);

	lbry_sha256_gpu_hash_20x2 <<<grid, block, 0, stream>>> (threads, (uint64_t*) d_Hash);
}

__host__
void lbry_sha256d_hash_20x2(int thr_id, uint32_t threads, uint32_t *d_Hash, hipStream_t stream)
{
	const int threadsperblock = 256;

	dim3 grid(threads/threadsperblock);
	dim3 block(threadsperblock);

	lbry_sha256d_gpu_hash_20x2 <<<grid, block, 64*4, stream>>> (threads, (uint64_t*) d_Hash);
}

__global__
__launch_bounds__(256,3)
void lbry_sha256d_gpu_hash_final(const uint32_t threads, const uint32_t startNonce, uint64_t *Hash512, uint32_t *resNonces)
{
	const uint32_t thread = (blockDim.x * blockIdx.x + threadIdx.x);
	if (thread < threads)
	{
		uint32_t __align__(8) buf[8]; // align for vectorize
		#pragma unroll
		for (int i=0;i<8;i++) buf[i] = c_H256[i];

		uint32_t* input = (uint32_t*) (&Hash512[thread * 8U]);

		uint32_t __align__(8) dat[16];
		#pragma unroll
		for (int i=0;i<5;i++) dat[i] = cuda_swab32(input[i]);
		#pragma unroll
		for (int i=0;i<5;i++) dat[i+5] = cuda_swab32(input[i+8]);
		dat[10] = 0x80000000;
		#pragma unroll
		for (int i=11;i<15;i++) dat[i] = 0;
		dat[15] = 0x140;

		sha256_round_body(dat, buf, c_K);

		// second sha256

		#pragma unroll
		for (int i=0;i<8;i++) dat[i] = buf[i];
		dat[8] = 0x80000000;
		#pragma unroll
		for (int i=9;i<15;i++) dat[i] = 0;
		dat[15] = 0x100;

		#pragma unroll
		for (int i=0;i<8;i++) buf[i] = c_H256[i];

		sha256_round_body(dat, buf, c_K);

		// valid nonces
		uint64_t high = cuda_swab32ll(((uint64_t*)buf)[3]);
		if (high <= d_target[0]) {
			// printf("%08x %08x - %016llx %016llx - %08x %08x\n", buf[7], buf[6], high, d_target[0], c_target[1], c_target[0]);
			uint32_t nonce = startNonce + thread;
			resNonces[1] = atomicExch(resNonces, nonce);
			d_target[0] = high;
		}
	}
}

__host__
void lbry_sha256d_hash_final(int thr_id, uint32_t threads, uint32_t startNonce, uint32_t *d_inputHash, uint32_t *resNonces, hipStream_t stream)
{
	const int threadsperblock = 256;

	dim3 grid(threads/threadsperblock);
	dim3 block(threadsperblock);

	CUDA_SAFE_CALL(hipMemset(d_resNonces, 0xFF, 2 * sizeof(uint32_t)));
	hipDeviceSynchronize();

	lbry_sha256d_gpu_hash_final <<<grid, block, 0, stream>>> (threads, startNonce, (uint64_t*) d_inputHash, d_resNonces);

	hipDeviceSynchronize();

	CUDA_SAFE_CALL(hipMemcpy(resNonces, d_resNonces, 2 * sizeof(uint32_t), hipMemcpyDeviceToHost));
	if (resNonces[0] == resNonces[1]) {
		resNonces[1] = UINT32_MAX;
	}
}