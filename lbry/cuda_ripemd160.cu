#include "hip/hip_runtime.h"
/*
 * ripemd-160 kernel implementation.
 *
 * ==========================(LICENSE BEGIN)============================
 *
 * Copyright (c) 2014, 2016  djm34, tpruvot
 *
 * Permission is hereby granted, free of charge, to any person obtaining
 * a copy of this software and associated documentation files (the
 * "Software"), to deal in the Software without restriction, including
 * without limitation the rights to use, copy, modify, merge, publish,
 * distribute, sublicense, and/or sell copies of the Software, and to
 * permit persons to whom the Software is furnished to do so, subject to
 * the following conditions:
 *
 * The above copyright notice and this permission notice shall be
 * included in all copies or substantial portions of the Software.
 *
 * THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND,
 * EXPRESS OR IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF
 * MERCHANTABILITY, FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT.
 * IN NO EVENT SHALL THE AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY
 * CLAIM, DAMAGES OR OTHER LIABILITY, WHETHER IN AN ACTION OF CONTRACT,
 * TORT OR OTHERWISE, ARISING FROM, OUT OF OR IN CONNECTION WITH THE
 * SOFTWARE OR THE USE OR OTHER DEALINGS IN THE SOFTWARE.
 *
 * ===========================(LICENSE END)=============================
 *
 */
#include <stdio.h>
#include <stdint.h>
#include <memory.h>

#include <cuda_helper.h>

static __constant__ uint32_t c_IV[5] = {
	0x67452301u, 0xEFCDAB89u, 0x98BADCFEu, 0x10325476u, 0xC3D2E1F0u
};

__device__ __forceinline__
uint32_t xor3b(const uint32_t a, const uint32_t b, const uint32_t c) {
	uint32_t result;
#if __CUDA_ARCH__ >= 500 && CUDA_VERSION >= 7050
	asm ("lop3.b32 %0, %1, %2, %3, 0x96; // xor3b"  //0x96 = 0xF0 ^ 0xCC ^ 0xAA
		: "=r"(result) : "r"(a), "r"(b),"r"(c));
#else
	result = a^b^c;
#endif
	return result;
}

//__host__
//uint64_t xornot64(uint64_t a, uint64_t b, uint64_t c) {
//	return c ^ (a | !b);
//}

__forceinline__ __device__
uint64_t xornot64(uint64_t a, uint64_t b, uint64_t c)
{
	uint64_t result;
	asm("{  .reg .u64 m,n; // xornot64\n\t"
		"not.b64 m,%2; \n\t"
		"or.b64 n, %1,m;\n\t"
		"xor.b64 %0, n,%3;\n\t"
		"}\n\t"
		: "=l"(result) : "l"(a), "l"(b), "l"(c));
	return result;
}

//__host__
//uint64_t xornt64(uint64_t a, uint64_t b, uint64_t c) {
//	return a ^ (b | !c);
//}

__device__ __forceinline__
uint64_t xornt64(uint64_t a, uint64_t b, uint64_t c)
{
	uint64_t result;
	asm("{  .reg .u64 m,n; // xornt64\n\t"
		"not.b64 m,%3; \n\t"
		"or.b64 n, %2,m;\n\t"
		"xor.b64 %0, %1,n;\n\t"
		"}\n\t"
		: "=l"(result) : "l"(a), "l"(b), "l"(c));
	return result;
}

/*
 * Round functions for RIPEMD-128 and RIPEMD-160.
 */
#if 1
#define F1(x, y, z)   ((x) ^ (y) ^ (z))
#define F2(x, y, z)   ((((y) ^ (z)) & (x)) ^ (z))
#define F3(x, y, z)   (((x) | ~(y)) ^ (z))
#define F4(x, y, z)   ((((x) ^ (y)) & (z)) ^ (y))
#define F5(x, y, z)   ((x) ^ ((y) | ~(z)))
#else
#define F1(x, y, z)   xor3b(x,y,z)
#define F2(x, y, z)   xandx(x,y,z)
#define F3(x, y, z)   xornot64(x,y,z)
#define F4(x, y, z)   xandx(z,x,y)
#define F5(x, y, z)   xornt64(x,y,z)
#endif

/*
 * Round constants for RIPEMD-160.
 */
#define K11 0x00000000u
#define K12 0x5A827999u
#define K13 0x6ED9EBA1u
#define K14 0x8F1BBCDCu
#define K15 0xA953FD4Eu

#define K21 0x50A28BE6u
#define K22 0x5C4DD124u
#define K23 0x6D703EF3u
#define K24 0x7A6D76E9u
#define K25 0x00000000u

#define RR(a, b, c, d, e, f, s, r, k) { \
	a = SPH_T32(ROTL32(SPH_T32(a + f(b, c, d) + r + k), s) + e); \
	c = ROTL32(c, 10); \
}

#define ROUND1(a, b, c, d, e, f, s, r, k) \
	RR(a ## 1, b ## 1, c ## 1, d ## 1, e ## 1, f, s, r, K1 ## k)

#define ROUND2(a, b, c, d, e, f, s, r, k) \
	RR(a ## 2, b ## 2, c ## 2, d ## 2, e ## 2, f, s, r, K2 ## k)

#define RIPEMD160_ROUND_BODY(in, h) { \
	uint32_t A1, B1, C1, D1, E1; \
	uint32_t A2, B2, C2, D2, E2; \
	uint32_t tmp; \
\
	A1 = A2 = h[0]; \
	B1 = B2 = h[1]; \
	C1 = C2 = h[2]; \
	D1 = D2 = h[3]; \
	E1 = E2 = h[4]; \
\
	ROUND1(A, B, C, D, E, F1, 11, in[ 0],  1); \
	ROUND1(E, A, B, C, D, F1, 14, in[ 1],  1); \
	ROUND1(D, E, A, B, C, F1, 15, in[ 2],  1); \
	ROUND1(C, D, E, A, B, F1, 12, in[ 3],  1); \
	ROUND1(B, C, D, E, A, F1,  5, in[ 4],  1); \
	ROUND1(A, B, C, D, E, F1,  8, in[ 5],  1); \
	ROUND1(E, A, B, C, D, F1,  7, in[ 6],  1); \
	ROUND1(D, E, A, B, C, F1,  9, in[ 7],  1); \
	ROUND1(C, D, E, A, B, F1, 11, in[ 8],  1); \
	ROUND1(B, C, D, E, A, F1, 13, in[ 9],  1); \
	ROUND1(A, B, C, D, E, F1, 14, in[10],  1); \
	ROUND1(E, A, B, C, D, F1, 15, in[11],  1); \
	ROUND1(D, E, A, B, C, F1,  6, in[12],  1); \
	ROUND1(C, D, E, A, B, F1,  7, in[13],  1); \
	ROUND1(B, C, D, E, A, F1,  9, in[14],  1); \
	ROUND1(A, B, C, D, E, F1,  8, in[15],  1); \
\
	ROUND1(E, A, B, C, D, F2,  7, in[ 7],  2); \
	ROUND1(D, E, A, B, C, F2,  6, in[ 4],  2); \
	ROUND1(C, D, E, A, B, F2,  8, in[13],  2); \
	ROUND1(B, C, D, E, A, F2, 13, in[ 1],  2); \
	ROUND1(A, B, C, D, E, F2, 11, in[10],  2); \
	ROUND1(E, A, B, C, D, F2,  9, in[ 6],  2); \
	ROUND1(D, E, A, B, C, F2,  7, in[15],  2); \
	ROUND1(C, D, E, A, B, F2, 15, in[ 3],  2); \
	ROUND1(B, C, D, E, A, F2,  7, in[12],  2); \
	ROUND1(A, B, C, D, E, F2, 12, in[ 0],  2); \
	ROUND1(E, A, B, C, D, F2, 15, in[ 9],  2); \
	ROUND1(D, E, A, B, C, F2,  9, in[ 5],  2); \
	ROUND1(C, D, E, A, B, F2, 11, in[ 2],  2); \
	ROUND1(B, C, D, E, A, F2,  7, in[14],  2); \
	ROUND1(A, B, C, D, E, F2, 13, in[11],  2); \
	ROUND1(E, A, B, C, D, F2, 12, in[ 8],  2); \
\
	ROUND1(D, E, A, B, C, F3, 11, in[ 3],  3); \
	ROUND1(C, D, E, A, B, F3, 13, in[10],  3); \
	ROUND1(B, C, D, E, A, F3,  6, in[14],  3); \
	ROUND1(A, B, C, D, E, F3,  7, in[ 4],  3); \
	ROUND1(E, A, B, C, D, F3, 14, in[ 9],  3); \
	ROUND1(D, E, A, B, C, F3,  9, in[15],  3); \
	ROUND1(C, D, E, A, B, F3, 13, in[ 8],  3); \
	ROUND1(B, C, D, E, A, F3, 15, in[ 1],  3); \
	ROUND1(A, B, C, D, E, F3, 14, in[ 2],  3); \
	ROUND1(E, A, B, C, D, F3,  8, in[ 7],  3); \
	ROUND1(D, E, A, B, C, F3, 13, in[ 0],  3); \
	ROUND1(C, D, E, A, B, F3,  6, in[ 6],  3); \
	ROUND1(B, C, D, E, A, F3,  5, in[13],  3); \
	ROUND1(A, B, C, D, E, F3, 12, in[11],  3); \
	ROUND1(E, A, B, C, D, F3,  7, in[ 5],  3); \
	ROUND1(D, E, A, B, C, F3,  5, in[12],  3); \
\
	ROUND1(C, D, E, A, B, F4, 11, in[ 1],  4); \
	ROUND1(B, C, D, E, A, F4, 12, in[ 9],  4); \
	ROUND1(A, B, C, D, E, F4, 14, in[11],  4); \
	ROUND1(E, A, B, C, D, F4, 15, in[10],  4); \
	ROUND1(D, E, A, B, C, F4, 14, in[ 0],  4); \
	ROUND1(C, D, E, A, B, F4, 15, in[ 8],  4); \
	ROUND1(B, C, D, E, A, F4,  9, in[12],  4); \
	ROUND1(A, B, C, D, E, F4,  8, in[ 4],  4); \
	ROUND1(E, A, B, C, D, F4,  9, in[13],  4); \
	ROUND1(D, E, A, B, C, F4, 14, in[ 3],  4); \
	ROUND1(C, D, E, A, B, F4,  5, in[ 7],  4); \
	ROUND1(B, C, D, E, A, F4,  6, in[15],  4); \
	ROUND1(A, B, C, D, E, F4,  8, in[14],  4); \
	ROUND1(E, A, B, C, D, F4,  6, in[ 5],  4); \
	ROUND1(D, E, A, B, C, F4,  5, in[ 6],  4); \
	ROUND1(C, D, E, A, B, F4, 12, in[ 2],  4); \
\
	ROUND1(B, C, D, E, A, F5,  9, in[ 4],  5); \
	ROUND1(A, B, C, D, E, F5, 15, in[ 0],  5); \
	ROUND1(E, A, B, C, D, F5,  5, in[ 5],  5); \
	ROUND1(D, E, A, B, C, F5, 11, in[ 9],  5); \
	ROUND1(C, D, E, A, B, F5,  6, in[ 7],  5); \
	ROUND1(B, C, D, E, A, F5,  8, in[12],  5); \
	ROUND1(A, B, C, D, E, F5, 13, in[ 2],  5); \
	ROUND1(E, A, B, C, D, F5, 12, in[10],  5); \
	ROUND1(D, E, A, B, C, F5,  5, in[14],  5); \
	ROUND1(C, D, E, A, B, F5, 12, in[ 1],  5); \
	ROUND1(B, C, D, E, A, F5, 13, in[ 3],  5); \
	ROUND1(A, B, C, D, E, F5, 14, in[ 8],  5); \
	ROUND1(E, A, B, C, D, F5, 11, in[11],  5); \
	ROUND1(D, E, A, B, C, F5,  8, in[ 6],  5); \
	ROUND1(C, D, E, A, B, F5,  5, in[15],  5); \
	ROUND1(B, C, D, E, A, F5,  6, in[13],  5); \
\
	ROUND2(A, B, C, D, E, F5,  8, in[ 5],  1); \
	ROUND2(E, A, B, C, D, F5,  9, in[14],  1); \
	ROUND2(D, E, A, B, C, F5,  9, in[ 7],  1); \
	ROUND2(C, D, E, A, B, F5, 11, in[ 0],  1); \
	ROUND2(B, C, D, E, A, F5, 13, in[ 9],  1); \
	ROUND2(A, B, C, D, E, F5, 15, in[ 2],  1); \
	ROUND2(E, A, B, C, D, F5, 15, in[11],  1); \
	ROUND2(D, E, A, B, C, F5,  5, in[ 4],  1); \
	ROUND2(C, D, E, A, B, F5,  7, in[13],  1); \
	ROUND2(B, C, D, E, A, F5,  7, in[ 6],  1); \
	ROUND2(A, B, C, D, E, F5,  8, in[15],  1); \
	ROUND2(E, A, B, C, D, F5, 11, in[ 8],  1); \
	ROUND2(D, E, A, B, C, F5, 14, in[ 1],  1); \
	ROUND2(C, D, E, A, B, F5, 14, in[10],  1); \
	ROUND2(B, C, D, E, A, F5, 12, in[ 3],  1); \
	ROUND2(A, B, C, D, E, F5,  6, in[12],  1); \
\
	ROUND2(E, A, B, C, D, F4,  9, in[ 6],  2); \
	ROUND2(D, E, A, B, C, F4, 13, in[11],  2); \
	ROUND2(C, D, E, A, B, F4, 15, in[ 3],  2); \
	ROUND2(B, C, D, E, A, F4,  7, in[ 7],  2); \
	ROUND2(A, B, C, D, E, F4, 12, in[ 0],  2); \
	ROUND2(E, A, B, C, D, F4,  8, in[13],  2); \
	ROUND2(D, E, A, B, C, F4,  9, in[ 5],  2); \
	ROUND2(C, D, E, A, B, F4, 11, in[10],  2); \
	ROUND2(B, C, D, E, A, F4,  7, in[14],  2); \
	ROUND2(A, B, C, D, E, F4,  7, in[15],  2); \
	ROUND2(E, A, B, C, D, F4, 12, in[ 8],  2); \
	ROUND2(D, E, A, B, C, F4,  7, in[12],  2); \
	ROUND2(C, D, E, A, B, F4,  6, in[ 4],  2); \
	ROUND2(B, C, D, E, A, F4, 15, in[ 9],  2); \
	ROUND2(A, B, C, D, E, F4, 13, in[ 1],  2); \
	ROUND2(E, A, B, C, D, F4, 11, in[ 2],  2); \
\
	ROUND2(D, E, A, B, C, F3,  9, in[15],  3); \
	ROUND2(C, D, E, A, B, F3,  7, in[ 5],  3); \
	ROUND2(B, C, D, E, A, F3, 15, in[ 1],  3); \
	ROUND2(A, B, C, D, E, F3, 11, in[ 3],  3); \
	ROUND2(E, A, B, C, D, F3,  8, in[ 7],  3); \
	ROUND2(D, E, A, B, C, F3,  6, in[14],  3); \
	ROUND2(C, D, E, A, B, F3,  6, in[ 6],  3); \
	ROUND2(B, C, D, E, A, F3, 14, in[ 9],  3); \
	ROUND2(A, B, C, D, E, F3, 12, in[11],  3); \
	ROUND2(E, A, B, C, D, F3, 13, in[ 8],  3); \
	ROUND2(D, E, A, B, C, F3,  5, in[12],  3); \
	ROUND2(C, D, E, A, B, F3, 14, in[ 2],  3); \
	ROUND2(B, C, D, E, A, F3, 13, in[10],  3); \
	ROUND2(A, B, C, D, E, F3, 13, in[ 0],  3); \
	ROUND2(E, A, B, C, D, F3,  7, in[ 4],  3); \
	ROUND2(D, E, A, B, C, F3,  5, in[13],  3); \
\
	ROUND2(C, D, E, A, B, F2, 15, in[ 8],  4); \
	ROUND2(B, C, D, E, A, F2,  5, in[ 6],  4); \
	ROUND2(A, B, C, D, E, F2,  8, in[ 4],  4); \
	ROUND2(E, A, B, C, D, F2, 11, in[ 1],  4); \
	ROUND2(D, E, A, B, C, F2, 14, in[ 3],  4); \
	ROUND2(C, D, E, A, B, F2, 14, in[11],  4); \
	ROUND2(B, C, D, E, A, F2,  6, in[15],  4); \
	ROUND2(A, B, C, D, E, F2, 14, in[ 0],  4); \
	ROUND2(E, A, B, C, D, F2,  6, in[ 5],  4); \
	ROUND2(D, E, A, B, C, F2,  9, in[12],  4); \
	ROUND2(C, D, E, A, B, F2, 12, in[ 2],  4); \
	ROUND2(B, C, D, E, A, F2,  9, in[13],  4); \
	ROUND2(A, B, C, D, E, F2, 12, in[ 9],  4); \
	ROUND2(E, A, B, C, D, F2,  5, in[ 7],  4); \
	ROUND2(D, E, A, B, C, F2, 15, in[10],  4); \
	ROUND2(C, D, E, A, B, F2,  8, in[14],  4); \
\
	ROUND2(B, C, D, E, A, F1,  8, in[12],  5); \
	ROUND2(A, B, C, D, E, F1,  5, in[15],  5); \
	ROUND2(E, A, B, C, D, F1, 12, in[10],  5); \
	ROUND2(D, E, A, B, C, F1,  9, in[ 4],  5); \
	ROUND2(C, D, E, A, B, F1, 12, in[ 1],  5); \
	ROUND2(B, C, D, E, A, F1,  5, in[ 5],  5); \
	ROUND2(A, B, C, D, E, F1, 14, in[ 8],  5); \
	ROUND2(E, A, B, C, D, F1,  6, in[ 7],  5); \
	ROUND2(D, E, A, B, C, F1,  8, in[ 6],  5); \
	ROUND2(C, D, E, A, B, F1, 13, in[ 2],  5); \
	ROUND2(B, C, D, E, A, F1,  6, in[13],  5); \
	ROUND2(A, B, C, D, E, F1,  5, in[14],  5); \
	ROUND2(E, A, B, C, D, F1, 15, in[ 0],  5); \
	ROUND2(D, E, A, B, C, F1, 13, in[ 3],  5); \
	ROUND2(C, D, E, A, B, F1, 11, in[ 9],  5); \
	ROUND2(B, C, D, E, A, F1, 11, in[11],  5); \
\
	tmp  = (h[1] + C1 + D2); \
	h[1] = (h[2] + D1 + E2); \
	h[2] = (h[3] + E1 + A2); \
	h[3] = (h[4] + A1 + B2); \
	h[4] = (h[0] + B1 + C2); \
	h[0] = tmp; \
}

__global__
void lbry_ripemd160_gpu_hash_32x2(const uint32_t threads, uint64_t *g_hash)
{
	const uint32_t thread = (blockDim.x * blockIdx.x + threadIdx.x);
	if (thread < threads)
	{
		uint32_t *hash = (uint32_t*) (&g_hash[thread * 8U]);

		uint32_t in[16];
		#pragma unroll
		for (int i=0; i<8; i++)
			in[i] = (hash[i]);
		in[8] = 0x80;

		#pragma unroll
		for (int i=9;i<16;i++) in[i] = 0;

		in[14] = 0x100; // size in bits

		uint32_t h[5];
		#pragma unroll
		for (int i=0; i<5; i++)
			h[i] = c_IV[i];

		RIPEMD160_ROUND_BODY(in, h);

		#pragma unroll
		for (int i=0; i<5; i++)
			hash[i] = h[i];

#ifdef PAD_ZEROS
		// 20 bytes hash on 32 output space
		hash[5] = 0;
		hash[6] = 0;
		hash[7] = 0;
#endif
		// second 32 bytes block hash
		hash += 8;

		#pragma unroll
		for (int i=0; i<8; i++)
			in[i] = (hash[i]);
		in[8] = 0x80;

		#pragma unroll
		for (int i=9;i<16;i++) in[i] = 0;

		in[14] = 0x100; // size in bits

		#pragma unroll
		for (int i=0; i<5; i++)
			h[i] = c_IV[i];

		RIPEMD160_ROUND_BODY(in, h);

		#pragma unroll
		for (int i=0; i<5; i++)
			hash[i] = h[i];

#ifdef PAD_ZEROS
		// 20 bytes hash on 32 output space
		hash[5] = 0;
		hash[6] = 0;
		hash[7] = 0;
#endif
	}
}

__host__
void lbry_ripemd160_hash_32x2(int thr_id, uint32_t threads, uint32_t *g_Hash, hipStream_t stream)
{
	const uint32_t threadsperblock = 128;

	dim3 grid(threads/threadsperblock);
	dim3 block(threadsperblock);

	lbry_ripemd160_gpu_hash_32x2 <<<grid, block, 0, stream>>> (threads, (uint64_t*) g_Hash);
}

void lbry_ripemd160_init(int thr_id)
{
	//hipMemcpyToSymbol(HIP_SYMBOL(c_IV), IV, sizeof(IV), 0, hipMemcpyHostToDevice);
}
