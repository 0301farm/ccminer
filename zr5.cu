#include "hip/hip_runtime.h"
/* Ziftrcoin ZR5 CUDA Implementation, (c) tpruvot 2015 */

extern "C" {
#include "sph/sph_blake.h"
#include "sph/sph_groestl.h"
#include "sph/sph_skein.h"
#include "sph/sph_jh.h"
#include "sph/sph_keccak.h"
}

#include "miner.h"
#include "cuda_helper.h"

#include <stdio.h>
#include <memory.h>

#define ZR_BLAKE   0
#define ZR_GROESTL 1
#define ZR_JH512   2
#define ZR_SKEIN   3

#define POK_BOOL_MASK 0x00008000
#define POK_DATA_MASK 0xFFFF0000

static uint32_t* d_hash[MAX_GPUS];
static uint16_t* d_pokh[MAX_GPUS];
static uint16_t* h_poks[MAX_GPUS];

static uint32_t* d_blake[MAX_GPUS];
static uint32_t* d_groes[MAX_GPUS];
static uint32_t* d_jh512[MAX_GPUS];
static uint32_t* d_skein[MAX_GPUS];

__constant__ uint8_t d_permut[24][4];
static const uint8_t permut[24][4] = {
	{0, 1, 2, 3},
	{0, 1, 3, 2},
	{0, 2, 1, 3},
	{0, 2, 3, 1},
	{0, 3, 1, 2},
	{0, 3, 2, 1},
	{1, 0, 2, 3},
	{1, 0, 3, 2},
	{1, 2, 0, 3},
	{1, 2, 3, 0},
	{1, 3, 0, 2},
	{1, 3, 2, 0},
	{2, 0, 1, 3},
	{2, 0, 3, 1},
	{2, 1, 0, 3},
	{2, 1, 3, 0},
	{2, 3, 0, 1},
	{2, 3, 1, 0},
	{3, 0, 1, 2},
	{3, 0, 2, 1},
	{3, 1, 0, 2},
	{3, 1, 2, 0},
	{3, 2, 0, 1},
	{3, 2, 1, 0}
};

// CPU HASH
extern "C" void zr5hash(void *output, const void *input)
{
	sph_keccak512_context ctx_keccak;
	sph_blake512_context ctx_blake;
	sph_groestl512_context ctx_groestl;
	sph_jh512_context ctx_jh;
	sph_skein512_context ctx_skein;

	uchar _ALIGN(64) hash[64];
	uint32_t *phash = (uint32_t *) hash;
	uint32_t norder;

	sph_keccak512_init(&ctx_keccak);
	sph_keccak512(&ctx_keccak, (const void*) input, 80);
	sph_keccak512_close(&ctx_keccak, (void*) phash);

	norder = phash[0] % ARRAY_SIZE(permut); /* % 24 */

	for(int i = 0; i < 4; i++)
	{
		switch (permut[norder][i]) {
		case ZR_BLAKE:
			sph_blake512_init(&ctx_blake);
			sph_blake512(&ctx_blake, (const void*) phash, 64);
			sph_blake512_close(&ctx_blake, phash);
			break;
		case ZR_GROESTL:
			sph_groestl512_init(&ctx_groestl);
			sph_groestl512(&ctx_groestl, (const void*) phash, 64);
			sph_groestl512_close(&ctx_groestl, phash);
			break;
		case ZR_JH512:
			sph_jh512_init(&ctx_jh);
			sph_jh512(&ctx_jh, (const void*) phash, 64);
			sph_jh512_close(&ctx_jh, phash);
			break;
		case ZR_SKEIN:
			sph_skein512_init(&ctx_skein);
			sph_skein512(&ctx_skein, (const void*) phash, 64);
			sph_skein512_close(&ctx_skein, phash);
			break;
		default:
			break;
		}
	}
	memcpy(output, phash, 32);
}

extern "C" void zr5hash_pok(void *output, uint32_t *pdata)
{
	const uint32_t version = pdata[0] & (~POK_DATA_MASK);
	uint32_t _ALIGN(64) hash[8];

	pdata[0] = version;
	zr5hash(hash, pdata);

	// fill PoK
	pdata[0] = version | (hash[0] & POK_DATA_MASK);
	zr5hash(hash, pdata);

	memcpy(output, hash, 32);
}

__global__
void zr5_copy_round_data_gpu(uint32_t threads, uint32_t *d_hash, uint32_t* d_blake, uint32_t* d_groes, uint32_t* d_jh512, uint32_t* d_skein, int rnd)
{
	// copy 64 bytes hash in the right algo buffer
	uint32_t thread = (blockDim.x * blockIdx.x + threadIdx.x);
	if (thread < threads)
	{
		const uint64_t offset = thread * 64 / 4;
		uint32_t *phash = &d_hash[offset];
		// algos hash order
		uint32_t norder = phash[0] % ARRAY_SIZE(permut);
		uint32_t algo = d_permut[norder][rnd];
		uint32_t* buffers[4] = { d_blake, d_groes, d_jh512, d_skein };

		if (rnd > 0) {
			int algosrc = d_permut[norder][rnd - 1];
			phash = buffers[algosrc] + offset;
		}

		// uint4 = 4x4 uint32_t = 16 bytes
		uint4 *psrc = (uint4*) phash;
		uint4 *pdst = (uint4*) (buffers[algo] + offset);
		pdst[0] = psrc[0];
		pdst[1] = psrc[1];
		pdst[2] = psrc[2];
		pdst[3] = psrc[3];
	}
}

__host__
void zr5_move_data_to_hash(int thr_id, uint32_t threads, int rnd)
{
	const uint32_t threadsperblock = 128;
	dim3 grid((threads + threadsperblock - 1) / threadsperblock);
	dim3 block(threadsperblock);

	zr5_copy_round_data_gpu <<<grid, block>>> (threads, d_hash[thr_id], d_blake[thr_id], d_groes[thr_id], d_jh512[thr_id], d_skein[thr_id], rnd);
}

__global__
void zr5_final_round_data_gpu(uint32_t threads, uint32_t* d_blake, uint32_t* d_groes, uint32_t* d_jh512, uint32_t* d_skein, uint32_t *d_hash, uint16_t *d_pokh)
{
	// after the 4 algos rounds, copy back hash to d_hash
	const uint32_t thread = (blockDim.x * blockIdx.x + threadIdx.x);
	if (thread < threads)
	{
		const uint64_t offset = thread * 16; // 64 / 4;
		uint32_t *phash = &d_hash[offset];
		uint16_t norder = phash[0] % ARRAY_SIZE(permut);
		uint16_t algosrc = d_permut[norder][3];

		uint32_t* buffers[4] = { d_blake, d_groes, d_jh512, d_skein };

		// copy only hash[0] + hash[6..7]
		uint2 *psrc = (uint2*) (buffers[algosrc] + offset);
		uint2 *pdst = (uint2*) phash;

		pdst[0].x = psrc[0].x;
		pdst[3] = psrc[3];

		//phash[7] = *(buffers[algosrc] + offset + 7);
	}
}

__host__
void zr5_final_round(int thr_id, uint32_t threads)
{
	const uint32_t threadsperblock = 128;
	dim3 grid((threads + threadsperblock - 1) / threadsperblock);
	dim3 block(threadsperblock);

	zr5_final_round_data_gpu <<<grid, block>>> (threads, d_blake[thr_id], d_groes[thr_id], d_jh512[thr_id], d_skein[thr_id], d_hash[thr_id], d_pokh[thr_id]);
}

extern void jackpot_keccak512_cpu_init(int thr_id, uint32_t threads);
extern void jackpot_keccak512_cpu_setBlock(void *pdata, size_t inlen);

extern void zr5_keccak512_cpu_hash(int thr_id, uint32_t threads, uint32_t startNounce, uint32_t *d_hash);
extern void zr5_keccak512_cpu_hash_pok(int thr_id, uint32_t threads, uint32_t startNounce, uint32_t* pdata, uint32_t *d_hash, uint16_t *d_poks);

extern void quark_blake512_cpu_init(int thr_id, uint32_t threads);
extern void quark_blake512_cpu_hash_64(int thr_id, uint32_t threads, uint32_t startNounce, uint32_t *d_nonceVector, uint32_t *d_hash, int order);

extern void quark_groestl512_cpu_init(int thr_id, uint32_t threads);
extern void quark_groestl512_cpu_hash_64(int thr_id, uint32_t threads, uint32_t startNounce, uint32_t *d_nonceVector, uint32_t *d_hash, int order);

extern void quark_jh512_cpu_init(int thr_id, uint32_t threads);
extern void quark_jh512_cpu_hash_64(int thr_id, uint32_t threads, uint32_t startNounce, uint32_t *d_nonceVector, uint32_t *d_hash, int order);

extern void quark_skein512_cpu_init(int thr_id, uint32_t threads);
extern void quark_skein512_cpu_hash_64(int thr_id, uint32_t threads, uint32_t startNounce, uint32_t *d_nonceVector, uint32_t *d_hash, int order);

static bool init[MAX_GPUS] = { 0 };

extern "C" int scanhash_zr5(int thr_id, uint32_t *pdata, const uint32_t *ptarget,
	uint32_t max_nonce, unsigned long *hashes_done)
{
	uint32_t _ALIGN(64) tmpdata[20];
	const uint32_t version = pdata[0] & (~POK_DATA_MASK);
	const uint32_t first_nonce = pdata[19];
	uint32_t throughput =  device_intensity(thr_id, __func__, 1U << 18);
	throughput = min(throughput, (1U << 20)-1024);
	throughput = min(throughput, max_nonce - first_nonce);

	if (opt_benchmark)
		((uint32_t*)ptarget)[7] = 0x0000ff;

	memcpy(tmpdata, pdata, 80);

	if (!init[thr_id])
	{
		hipSetDevice(device_map[thr_id]);

		// hash buffer = keccak hash 64 required
		hipMalloc(&d_hash[thr_id], 64 * throughput);
		hipMalloc(&d_pokh[thr_id], 2 * throughput);

		hipMemcpyToSymbol(HIP_SYMBOL(d_permut), permut, 24*4, 0, hipMemcpyHostToDevice);
		hipHostMalloc(&h_poks[thr_id], 2 * throughput);

		// data buffers for the 4 rounds
		hipMalloc(&d_blake[thr_id], 64 * throughput);
		hipMalloc(&d_groes[thr_id], 64 * throughput);
		hipMalloc(&d_jh512[thr_id], 64 * throughput);
		hipMalloc(&d_skein[thr_id], 64 * throughput);

		jackpot_keccak512_cpu_init(thr_id, throughput);

		quark_blake512_cpu_init(thr_id, throughput);
		quark_groestl512_cpu_init(thr_id, throughput);
		quark_jh512_cpu_init(thr_id, throughput);
		quark_skein512_cpu_init(thr_id, throughput);

		cuda_check_cpu_init(thr_id, throughput);

		CUDA_SAFE_CALL(hipDeviceSynchronize());

		init[thr_id] = true;
	}

	tmpdata[0] = version;
	jackpot_keccak512_cpu_setBlock((void*)tmpdata, 80);
	cuda_check_cpu_setTarget(ptarget);

	do {
		int order = 0;

		// Keccak512 Hash with CUDA
		zr5_keccak512_cpu_hash(thr_id, throughput, pdata[19], d_hash[thr_id]);

		for (int rnd=0; rnd<4; rnd++) {
			zr5_move_data_to_hash(thr_id, throughput, rnd);
			quark_blake512_cpu_hash_64(thr_id, throughput, pdata[19], NULL, d_blake[thr_id], order++);
			quark_groestl512_cpu_hash_64(thr_id, throughput, pdata[19], NULL, d_groes[thr_id], order++);
			quark_jh512_cpu_hash_64(thr_id, throughput, pdata[19], NULL, d_jh512[thr_id], order++);
			quark_skein512_cpu_hash_64(thr_id, throughput, pdata[19], NULL, d_skein[thr_id], order++);
		}

		// This generates all pok prefixes
		zr5_final_round(thr_id, throughput);

		// Keccak512 pok
		zr5_keccak512_cpu_hash_pok(thr_id, throughput, pdata[19], pdata, d_hash[thr_id], d_pokh[thr_id]);

		for (int rnd=0; rnd<4; rnd++) {
			zr5_move_data_to_hash(thr_id, throughput, rnd);
			quark_blake512_cpu_hash_64(thr_id, throughput, pdata[19], NULL, d_blake[thr_id], order++);
			quark_groestl512_cpu_hash_64(thr_id, throughput, pdata[19], NULL, d_groes[thr_id], order++);
			quark_jh512_cpu_hash_64(thr_id, throughput, pdata[19], NULL, d_jh512[thr_id], order++);
			quark_skein512_cpu_hash_64(thr_id, throughput, pdata[19], NULL, d_skein[thr_id], order++);
		}
		zr5_final_round(thr_id, throughput);

		uint32_t foundNonce = cuda_check_hash(thr_id, throughput, pdata[19], d_hash[thr_id]);
		if (foundNonce != UINT32_MAX)
		{
			uint32_t vhash64[8];
			uint32_t oldp0 = pdata[0];
			uint32_t oldp19 = pdata[19];
			uint32_t offset = foundNonce - pdata[19];
			uint32_t pok = 0;

			*hashes_done = pdata[19] - first_nonce + throughput;

			hipMemcpy(h_poks[thr_id], d_pokh[thr_id], 2 * throughput, hipMemcpyDeviceToHost);
			pok = version | (0x10000UL * h_poks[thr_id][offset]);
			pdata[0] = pok; pdata[19] = foundNonce;
			zr5hash(vhash64, pdata);
			if (vhash64[7] <= ptarget[7] && fulltest(vhash64, ptarget)) {
				int res = 1;
				uint32_t secNonce = cuda_check_hash_suppl(thr_id, throughput, pdata[19], d_hash[thr_id], 1);
				if (secNonce != 0) {
					offset = secNonce - oldp19;
					pok = version | (0x10000UL * h_poks[thr_id][offset]);
					memcpy(tmpdata, pdata, 80);
					tmpdata[0] = pok; tmpdata[19] = secNonce;
					zr5hash(vhash64, tmpdata);
					if (vhash64[7] <= ptarget[7] && fulltest(vhash64, ptarget)) {
						pdata[21] = secNonce;
						pdata[22] = pok;
						res++;
					}
				}
				return res;
			} else {
				applog(LOG_WARNING, "GPU #%d: result for %08x does not validate on CPU!", thr_id, foundNonce);
				pdata[19]++;
				pdata[0] = oldp0;
			}
		} else
			pdata[19] += throughput;

	} while (pdata[19] < max_nonce && !work_restart[thr_id].restart);

	*hashes_done = pdata[19] - first_nonce + 1;
	return 0;
}
