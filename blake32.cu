#include "hip/hip_runtime.h"
/**
 * Blake-256 Cuda Kernel (Tested on SM 5.0)
 *
 * Tanguy Pruvot - Aug. 2014
 */

#include "miner.h"

extern "C" {
#include "sph/sph_blake.h"
#include <stdint.h>
#include <memory.h>
}

/* threads per block */
#define TPB 128

extern "C" int blake256_rounds = 14;

/* hash by cpu with blake 256 */
extern "C" void blake256hash(void *output, const void *input, int rounds = 14)
{
	unsigned char hash[64];
	sph_blake256_context ctx;

	/* in sph_blake.c */
	blake256_rounds = rounds;

	sph_blake256_init(&ctx);
	sph_blake256(&ctx, input, 80);
	sph_blake256_close(&ctx, hash);
	memcpy(output, hash, 32);
}

#include "cuda_helper.h"

#define MAXU 0xffffffffU

// in cpu-miner.c
extern bool opt_n_threads;
extern bool opt_benchmark;
extern int device_map[8];

uint32_t crc32(const uint32_t *buf, size_t size);

__constant__
static uint32_t __align__(32) c_Target[8];

__constant__
static uint32_t __align__(32) c_data[20];

static uint32_t *d_resNounce[8];
static uint32_t *h_resNounce[8];
static uint32_t extra_results[2] = { MAXU, MAXU };

#define USE_CACHE 1
#if USE_CACHE
__device__
static uint32_t cache[8];
__device__
static uint32_t prevsum = 0;
#endif

/* prefer uint32_t to prevent size conversions = speed +5/10 % */
__constant__
static uint32_t __align__(32) c_sigma[16][16];
const uint32_t host_sigma[16][16] = {
	{ 0, 1, 2, 3, 4, 5, 6, 7, 8, 9, 10, 11, 12, 13, 14, 15 },
	{14, 10, 4, 8, 9, 15, 13, 6, 1, 12, 0, 2, 11, 7, 5, 3 },
	{11, 8, 12, 0, 5, 2, 15, 13, 10, 14, 3, 6, 7, 1, 9, 4 },
	{ 7, 9, 3, 1, 13, 12, 11, 14, 2, 6, 5, 10, 4, 0, 15, 8 },
	{ 9, 0, 5, 7, 2, 4, 10, 15, 14, 1, 11, 12, 6, 8, 3, 13 },
	{ 2, 12, 6, 10, 0, 11, 8, 3, 4, 13, 7, 5, 15, 14, 1, 9 },
	{12, 5, 1, 15, 14, 13, 4, 10, 0, 7, 6, 3, 9, 2, 8, 11 },
	{13, 11, 7, 14, 12, 1, 3, 9, 5, 0, 15, 4, 8, 6, 2, 10 },
	{ 6, 15, 14, 9, 11, 3, 0, 8, 12, 2, 13, 7, 1, 4, 10, 5 },
	{10, 2, 8, 4, 7, 6, 1, 5, 15, 11, 9, 14, 3, 12, 13 , 0 },
	{ 0, 1, 2, 3, 4, 5, 6, 7, 8, 9, 10, 11, 12, 13, 14, 15 },
	{14, 10, 4, 8, 9, 15, 13, 6, 1, 12, 0, 2, 11, 7, 5, 3 },
	{11, 8, 12, 0, 5, 2, 15, 13, 10, 14, 3, 6, 7, 1, 9, 4 },
	{ 7, 9, 3, 1, 13, 12, 11, 14, 2, 6, 5, 10, 4, 0, 15, 8 },
	{ 9, 0, 5, 7, 2, 4, 10, 15, 14, 1, 11, 12, 6, 8, 3, 13 },
	{ 2, 12, 6, 10, 0, 11, 8, 3, 4, 13, 7, 5, 15, 14, 1, 9 }
};

__device__ __constant__
static const uint32_t __align__(32) c_IV256[8] = {
	SPH_C32(0x6A09E667), SPH_C32(0xBB67AE85),
	SPH_C32(0x3C6EF372), SPH_C32(0xA54FF53A),
	SPH_C32(0x510E527F), SPH_C32(0x9B05688C),
	SPH_C32(0x1F83D9AB), SPH_C32(0x5BE0CD19)
};

__device__ __constant__
static const uint32_t __align__(32) c_u256[16] = {
	SPH_C32(0x243F6A88), SPH_C32(0x85A308D3),
	SPH_C32(0x13198A2E), SPH_C32(0x03707344),
	SPH_C32(0xA4093822), SPH_C32(0x299F31D0),
	SPH_C32(0x082EFA98), SPH_C32(0xEC4E6C89),
	SPH_C32(0x452821E6), SPH_C32(0x38D01377),
	SPH_C32(0xBE5466CF), SPH_C32(0x34E90C6C),
	SPH_C32(0xC0AC29B7), SPH_C32(0xC97C50DD),
	SPH_C32(0x3F84D5B5), SPH_C32(0xB5470917)
};

#if 0
#define GS(m0, m1, c0, c1, a, b, c, d)   do { \
		a = SPH_T32(a + b + (m0 ^ c1)); \
		d = SPH_ROTR32(d ^ a, 16); \
		c = SPH_T32(c + d); \
		b = SPH_ROTR32(b ^ c, 12); \
		a = SPH_T32(a + b + (m1 ^ c0)); \
		d = SPH_ROTR32(d ^ a, 8); \
		c = SPH_T32(c + d); \
		b = SPH_ROTR32(b ^ c, 7); \
	} while (0)

#define ROUND_S(r)   do { \
	GS(Mx(r, 0x0), Mx(r, 0x1), CSx(r, 0x0), CSx(r, 0x1), v[0], v[4], v[0x8], v[0xC]); \
	GS(Mx(r, 0x2), Mx(r, 0x3), CSx(r, 0x2), CSx(r, 0x3), v[1], v[5], v[0x9], v[0xD]); \
	GS(Mx(r, 0x4), Mx(r, 0x5), CSx(r, 0x4), CSx(r, 0x5), v[2], v[6], v[0xA], v[0xE]); \
	GS(Mx(r, 0x6), Mx(r, 0x7), CSx(r, 0x6), CSx(r, 0x7), v[3], v[7], v[0xB], v[0xF]); \
	GS(Mx(r, 0x8), Mx(r, 0x9), CSx(r, 0x8), CSx(r, 0x9), v[0], v[5], v[0xA], v[0xF]); \
	GS(Mx(r, 0xA), Mx(r, 0xB), CSx(r, 0xA), CSx(r, 0xB), v[1], v[6], v[0xB], v[0xC]); \
	GS(Mx(r, 0xC), Mx(r, 0xD), CSx(r, 0xC), CSx(r, 0xD), v[2], v[7], v[0x8], v[0xD]); \
	GS(Mx(r, 0xE), Mx(r, 0xF), CSx(r, 0xE), CSx(r, 0xF), v[3], v[4], v[0x9], v[0xE]); \
} while (0)
#endif

#define GS(a,b,c,d,x) { \
	const uint32_t idx1 = c_sigma[i][x]; \
	const uint32_t idx2 = c_sigma[i][x+1]; \
	v[a] += (m[idx1] ^ c_u256[idx2]) + v[b]; \
	v[d] = SPH_ROTL32(v[d] ^ v[a], 16); \
	v[c] += v[d]; \
	v[b] = SPH_ROTR32(v[b] ^ v[c], 12); \
\
	v[a] += (m[idx2] ^ c_u256[idx1]) + v[b]; \
	v[d] = SPH_ROTR32(v[d] ^ v[a], 8); \
	v[c] += v[d]; \
	v[b] = SPH_ROTR32(v[b] ^ v[c], 7); \
}

/* Second part (64-80) msg never change, store it */
__device__ __constant__
static const uint32_t __align__(32) c_Padding[16] = {
	0, 0, 0, 0,
	0x80000000UL, 0, 0, 0,
	0, 0, 0, 0,
	0, 1, 0, 640,
};

__device__ static
void blake256_compress(uint32_t *h, const uint32_t *block, const uint32_t T0, int blakerounds)
{
	uint32_t /* __align__(8) */ m[16];
	uint32_t /* __align__(8) */ v[16];

	m[0] = block[0];
	m[1] = block[1];
	m[2] = block[2];
	m[3] = block[3];

	for (uint32_t i = 4; i < 16; i++) {
		m[i] = (T0 == 0x200) ? block[i] : c_Padding[i];
	}

	//#pragma unroll 8
	for(uint32_t i = 0; i < 8; i++)
		v[i] = h[i];

	v[ 8] = c_u256[0];
	v[ 9] = c_u256[1];
	v[10] = c_u256[2];
	v[11] = c_u256[3];

	v[12] = c_u256[4] ^ T0;
	v[13] = c_u256[5] ^ T0;
	v[14] = c_u256[6];
	v[15] = c_u256[7];

	for (int i = 0; i < blakerounds; i++) {
		/* column step */
		GS(0, 4, 0x8, 0xC, 0x0);
		GS(1, 5, 0x9, 0xD, 0x2);
		GS(2, 6, 0xA, 0xE, 0x4);
		GS(3, 7, 0xB, 0xF, 0x6);
		/* diagonal step */
		GS(0, 5, 0xA, 0xF, 0x8);
		GS(1, 6, 0xB, 0xC, 0xA);
		GS(2, 7, 0x8, 0xD, 0xC);
		GS(3, 4, 0x9, 0xE, 0xE);
	}

	//#pragma unroll 16
	for (uint32_t i = 0; i < 16; i++) {
		uint32_t j = i % 8;
		h[j] ^= v[i];
	}
}

__global__
void blake256_gpu_hash_80(uint32_t threads, uint32_t startNounce, uint32_t *resNounce, const int blakerounds, const int crcsum)
{
	uint32_t thread = (blockDim.x * blockIdx.x + threadIdx.x);
	if (thread < threads)
	{
		const uint32_t nounce = startNounce + thread;
		uint32_t h[8];

		#pragma unroll
		for(int i=0; i<8; i++) {
			h[i] = c_IV256[i];
		}

#if !USE_CACHE
		blake256_compress(h, c_data, 512, blakerounds);
#else
		if (crcsum != prevsum) {
			prevsum = crcsum;
			blake256_compress(h, c_data, 512, blakerounds);
			#pragma unroll
			for(int i=0; i<8; i++) {
				cache[i] = h[i];
			}
		} else {
			#pragma unroll
			for(int i=0; i<8; i++) {
				h[i] = cache[i];
			}
		}
#endif
		// ------ Close: Bytes 64 to 80 ------ 

		uint32_t ending[4];
		ending[0] = c_data[16];
		ending[1] = c_data[17];
		ending[2] = c_data[18];
		ending[3] = nounce; /* our tested value */

		blake256_compress(h, ending, 640, blakerounds);

		for (int i = 7; i >= 0; i--) {
			uint32_t hash = cuda_swab32(h[i]);
			if (hash > c_Target[i]) {
				return;
			}
			if (hash < c_Target[i]) {
				break;
			}
		}

		/* keep the smallest nounce, + extra one if found */
		if (resNounce[0] > nounce) {
			resNounce[1] = resNounce[0];
			resNounce[0] = nounce;
		}
		else
			resNounce[1] = nounce;
	}
}

__host__
uint32_t blake256_cpu_hash_80(int thr_id, uint32_t threads, uint32_t startNounce, const int blakerounds, const uint32_t crcsum)
{
	const int threadsperblock = TPB;
	uint32_t result = MAXU;

	dim3 grid((threads + threadsperblock-1)/threadsperblock);
	dim3 block(threadsperblock);
	size_t shared_size = 0;

	/* Check error on Ctrl+C or kill to prevent segfaults on exit */
	if (hipMemset(d_resNounce[thr_id], 0xff, 2*sizeof(uint32_t)) != hipSuccess)
		return result;

	blake256_gpu_hash_80<<<grid, block, shared_size>>>(threads, startNounce, d_resNounce[thr_id], blakerounds, crcsum);
	hipDeviceSynchronize();
	if (hipSuccess == hipMemcpy(h_resNounce[thr_id], d_resNounce[thr_id], 2*sizeof(uint32_t), hipMemcpyDeviceToHost)) {
		hipDeviceSynchronize();
		result = h_resNounce[thr_id][0];
		extra_results[0] = h_resNounce[thr_id][1];
	}
	return result;
}

__host__
void blake256_cpu_setBlock_80(uint32_t *pdata, const uint32_t *ptarget)
{
	uint32_t data[20];
	memcpy(data, pdata, 80);
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(c_data), data, sizeof(data), 0, hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(c_sigma), host_sigma, sizeof(host_sigma), 0, hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(c_Target), ptarget, 32, 0, hipMemcpyHostToDevice));
}

extern "C" int scanhash_blake256(int thr_id, uint32_t *pdata, const uint32_t *ptarget,
	uint32_t max_nonce, unsigned long *hashes_done, uint32_t blakerounds=14)
{
	const uint32_t first_nonce = pdata[19];
	static bool init[8] = { 0, 0, 0, 0, 0, 0, 0, 0 };
	uint32_t throughput = min(TPB * 4096, max_nonce - first_nonce);
	uint32_t crcsum = MAXU;
	int rc = 0;

	if (extra_results[0] != MAXU) {
		// possible extra result found in previous call
		if (first_nonce <= extra_results[0] && max_nonce >= extra_results[0]) {
			pdata[19] = extra_results[0];
			*hashes_done = pdata[19] - first_nonce + 1;
			extra_results[0] = MAXU;
			rc = 1;
			goto exit_scan;
		}
	}

	if (opt_benchmark)
		((uint32_t*)ptarget)[7] = 0x00000f;

	if (!init[thr_id]) {
		if (opt_n_threads > 1) {
			CUDA_SAFE_CALL(hipSetDevice(device_map[thr_id]));
		}
		CUDA_SAFE_CALL(hipHostMalloc(&h_resNounce[thr_id], 2*sizeof(uint32_t)));
		CUDA_SAFE_CALL(hipMalloc(&d_resNounce[thr_id], 2*sizeof(uint32_t)));
		init[thr_id] = true;
	}

	if (opt_debug && throughput < (TPB * 4096))
		applog(LOG_DEBUG, "throughput=%u, start=%x, max=%x", throughput, first_nonce, max_nonce);

	blake256_cpu_setBlock_80(pdata, ptarget);
#if USE_CACHE
	crcsum = crc32(pdata, 64);
#endif

	do {
		// GPU HASH
		uint32_t foundNonce = blake256_cpu_hash_80(thr_id, throughput, pdata[19], blakerounds, crcsum);
		if (foundNonce != MAXU)
		{
			uint32_t endiandata[20];
			uint32_t vhashcpu[8];
			uint32_t Htarg = ptarget[7];

			for (int k=0; k < 19; k++)
				be32enc(&endiandata[k], pdata[k]);

			be32enc(&endiandata[19], foundNonce);

			blake256hash(vhashcpu, endiandata, blakerounds);

			if (vhashcpu[7] <= Htarg && fulltest(vhashcpu, ptarget))
			{
				pdata[19] = foundNonce;
				rc = 1;

				if (extra_results[0] != MAXU) {
					// Rare but possible if the throughput is big
					be32enc(&endiandata[19], extra_results[0]);
					blake256hash(vhashcpu, endiandata, blakerounds);
					if (vhashcpu[7] <= Htarg && fulltest(vhashcpu, ptarget)) {
						applog(LOG_NOTICE, "GPU found more than one result yippee!");
					} else {
						extra_results[0] = MAXU;
					}
				}

				goto exit_scan;
			}
			else if (vhashcpu[7] > Htarg) {
				applog(LOG_WARNING, "GPU #%d: result for nounce %08x is not in range: %x > %x", thr_id, foundNonce, vhashcpu[7], Htarg);
			}
			else if (vhashcpu[6] > ptarget[6]) {
				applog(LOG_WARNING, "GPU #%d: hash[6] for nounce %08x is not in range: %x > %x", thr_id, foundNonce, vhashcpu[6], ptarget[6]);
			}
			else {
				applog(LOG_WARNING, "GPU #%d: result for nounce %08x does not validate on CPU!", thr_id, foundNonce);
			}
		}

		pdata[19] += throughput;

	} while (pdata[19] < max_nonce && !work_restart[thr_id].restart);

exit_scan:
	*hashes_done = pdata[19] - first_nonce + 1;
#if 0
	/* reset the device to allow multiple instances
	 * could be made in cpu-miner... check later if required */
	if (opt_n_threads == 1) {
		CUDA_SAFE_CALL(hipDeviceReset());
		init[thr_id] = false;
	}
#endif
	// wait proper end of all threads
	hipDeviceSynchronize();
	return rc;
}

static uint32_t crc32_tab[] = {
	0x00000000, 0x77073096, 0xee0e612c, 0x990951ba, 0x076dc419, 0x706af48f,
	0xe963a535, 0x9e6495a3,	0x0edb8832, 0x79dcb8a4, 0xe0d5e91e, 0x97d2d988,
	0x09b64c2b, 0x7eb17cbd, 0xe7b82d07, 0x90bf1d91, 0x1db71064, 0x6ab020f2,
	0xf3b97148, 0x84be41de,	0x1adad47d, 0x6ddde4eb, 0xf4d4b551, 0x83d385c7,
	0x136c9856, 0x646ba8c0, 0xfd62f97a, 0x8a65c9ec,	0x14015c4f, 0x63066cd9,
	0xfa0f3d63, 0x8d080df5,	0x3b6e20c8, 0x4c69105e, 0xd56041e4, 0xa2677172,
	0x3c03e4d1, 0x4b04d447, 0xd20d85fd, 0xa50ab56b,	0x35b5a8fa, 0x42b2986c,
	0xdbbbc9d6, 0xacbcf940,	0x32d86ce3, 0x45df5c75, 0xdcd60dcf, 0xabd13d59,
	0x26d930ac, 0x51de003a, 0xc8d75180, 0xbfd06116, 0x21b4f4b5, 0x56b3c423,
	0xcfba9599, 0xb8bda50f, 0x2802b89e, 0x5f058808, 0xc60cd9b2, 0xb10be924,
	0x2f6f7c87, 0x58684c11, 0xc1611dab, 0xb6662d3d,	0x76dc4190, 0x01db7106,
	0x98d220bc, 0xefd5102a, 0x71b18589, 0x06b6b51f, 0x9fbfe4a5, 0xe8b8d433,
	0x7807c9a2, 0x0f00f934, 0x9609a88e, 0xe10e9818, 0x7f6a0dbb, 0x086d3d2d,
	0x91646c97, 0xe6635c01, 0x6b6b51f4, 0x1c6c6162, 0x856530d8, 0xf262004e,
	0x6c0695ed, 0x1b01a57b, 0x8208f4c1, 0xf50fc457, 0x65b0d9c6, 0x12b7e950,
	0x8bbeb8ea, 0xfcb9887c, 0x62dd1ddf, 0x15da2d49, 0x8cd37cf3, 0xfbd44c65,
	0x4db26158, 0x3ab551ce, 0xa3bc0074, 0xd4bb30e2, 0x4adfa541, 0x3dd895d7,
	0xa4d1c46d, 0xd3d6f4fb, 0x4369e96a, 0x346ed9fc, 0xad678846, 0xda60b8d0,
	0x44042d73, 0x33031de5, 0xaa0a4c5f, 0xdd0d7cc9, 0x5005713c, 0x270241aa,
	0xbe0b1010, 0xc90c2086, 0x5768b525, 0x206f85b3, 0xb966d409, 0xce61e49f,
	0x5edef90e, 0x29d9c998, 0xb0d09822, 0xc7d7a8b4, 0x59b33d17, 0x2eb40d81,
	0xb7bd5c3b, 0xc0ba6cad, 0xedb88320, 0x9abfb3b6, 0x03b6e20c, 0x74b1d29a,
	0xead54739, 0x9dd277af, 0x04db2615, 0x73dc1683, 0xe3630b12, 0x94643b84,
	0x0d6d6a3e, 0x7a6a5aa8, 0xe40ecf0b, 0x9309ff9d, 0x0a00ae27, 0x7d079eb1,
	0xf00f9344, 0x8708a3d2, 0x1e01f268, 0x6906c2fe, 0xf762575d, 0x806567cb,
	0x196c3671, 0x6e6b06e7, 0xfed41b76, 0x89d32be0, 0x10da7a5a, 0x67dd4acc,
	0xf9b9df6f, 0x8ebeeff9, 0x17b7be43, 0x60b08ed5, 0xd6d6a3e8, 0xa1d1937e,
	0x38d8c2c4, 0x4fdff252, 0xd1bb67f1, 0xa6bc5767, 0x3fb506dd, 0x48b2364b,
	0xd80d2bda, 0xaf0a1b4c, 0x36034af6, 0x41047a60, 0xdf60efc3, 0xa867df55,
	0x316e8eef, 0x4669be79, 0xcb61b38c, 0xbc66831a, 0x256fd2a0, 0x5268e236,
	0xcc0c7795, 0xbb0b4703, 0x220216b9, 0x5505262f, 0xc5ba3bbe, 0xb2bd0b28,
	0x2bb45a92, 0x5cb36a04, 0xc2d7ffa7, 0xb5d0cf31, 0x2cd99e8b, 0x5bdeae1d,
	0x9b64c2b0, 0xec63f226, 0x756aa39c, 0x026d930a, 0x9c0906a9, 0xeb0e363f,
	0x72076785, 0x05005713, 0x95bf4a82, 0xe2b87a14, 0x7bb12bae, 0x0cb61b38,
	0x92d28e9b, 0xe5d5be0d, 0x7cdcefb7, 0x0bdbdf21, 0x86d3d2d4, 0xf1d4e242,
	0x68ddb3f8, 0x1fda836e, 0x81be16cd, 0xf6b9265b, 0x6fb077e1, 0x18b74777,
	0x88085ae6, 0xff0f6a70, 0x66063bca, 0x11010b5c, 0x8f659eff, 0xf862ae69,
	0x616bffd3, 0x166ccf45, 0xa00ae278, 0xd70dd2ee, 0x4e048354, 0x3903b3c2,
	0xa7672661, 0xd06016f7, 0x4969474d, 0x3e6e77db, 0xaed16a4a, 0xd9d65adc,
	0x40df0b66, 0x37d83bf0, 0xa9bcae53, 0xdebb9ec5, 0x47b2cf7f, 0x30b5ffe9,
	0xbdbdf21c, 0xcabac28a, 0x53b39330, 0x24b4a3a6, 0xbad03605, 0xcdd70693,
	0x54de5729, 0x23d967bf, 0xb3667a2e, 0xc4614ab8, 0x5d681b02, 0x2a6f2b94,
	0xb40bbe37, 0xc30c8ea1, 0x5a05df1b, 0x2d02ef8d
};

uint32_t crc32(const uint32_t *buf, size_t size)
{
	const uint8_t *p;
	uint32_t crc = 0;

	p = (uint8_t *) buf;
	crc = crc ^ ~0U;

	while (size--)
		crc = crc32_tab[(crc ^ *p++) & 0xFF] ^ (crc >> 8);

	return crc ^ ~0U;
}