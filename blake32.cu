#include "hip/hip_runtime.h"
/**
 * Blake-256 Cuda Kernel (Tested on SM 5.0)
 *
 * Tanguy Pruvot - Aug. 2014
 */

#include "miner.h"

extern "C" {
#include "sph/sph_blake.h"
#include <stdint.h>
#include <memory.h>
}

/* threads per block */
#define TPB 128

/* crc32.c */
extern "C" uint32_t crc32_u32t(const uint32_t *buf, size_t size);

extern "C" int blake256_rounds = 14;

/* hash by cpu with blake 256 */
extern "C" void blake256hash(void *output, const void *input, int rounds = 14)
{
	unsigned char hash[64];
	sph_blake256_context ctx;

	/* in sph_blake.c */
	blake256_rounds = rounds;

	sph_blake256_init(&ctx);
	sph_blake256(&ctx, input, 80);
	sph_blake256_close(&ctx, hash);
	memcpy(output, hash, 32);
}

#include "cuda_helper.h"

#define MAXU 0xffffffffU

// in cpu-miner.c
extern bool opt_n_threads;
extern bool opt_benchmark;
extern int device_map[8];

__constant__
static uint32_t __align__(32) c_Target[8];

__constant__
static uint32_t __align__(32) c_data[20];

static uint32_t *d_resNounce[8];
static uint32_t *h_resNounce[8];
static uint32_t extra_results[2] = { MAXU, MAXU };

#define USE_CACHE 1
#if USE_CACHE
__device__
static uint32_t cache[8];
__device__
static uint32_t prevsum = 0;
#endif

/* prefer uint32_t to prevent size conversions = speed +5/10 % */
__constant__
static uint32_t __align__(32) c_sigma[16][16];
const uint32_t host_sigma[16][16] = {
	{ 0, 1, 2, 3, 4, 5, 6, 7, 8, 9, 10, 11, 12, 13, 14, 15 },
	{14, 10, 4, 8, 9, 15, 13, 6, 1, 12, 0, 2, 11, 7, 5, 3 },
	{11, 8, 12, 0, 5, 2, 15, 13, 10, 14, 3, 6, 7, 1, 9, 4 },
	{ 7, 9, 3, 1, 13, 12, 11, 14, 2, 6, 5, 10, 4, 0, 15, 8 },
	{ 9, 0, 5, 7, 2, 4, 10, 15, 14, 1, 11, 12, 6, 8, 3, 13 },
	{ 2, 12, 6, 10, 0, 11, 8, 3, 4, 13, 7, 5, 15, 14, 1, 9 },
	{12, 5, 1, 15, 14, 13, 4, 10, 0, 7, 6, 3, 9, 2, 8, 11 },
	{13, 11, 7, 14, 12, 1, 3, 9, 5, 0, 15, 4, 8, 6, 2, 10 },
	{ 6, 15, 14, 9, 11, 3, 0, 8, 12, 2, 13, 7, 1, 4, 10, 5 },
	{10, 2, 8, 4, 7, 6, 1, 5, 15, 11, 9, 14, 3, 12, 13 , 0 },
	{ 0, 1, 2, 3, 4, 5, 6, 7, 8, 9, 10, 11, 12, 13, 14, 15 },
	{14, 10, 4, 8, 9, 15, 13, 6, 1, 12, 0, 2, 11, 7, 5, 3 },
	{11, 8, 12, 0, 5, 2, 15, 13, 10, 14, 3, 6, 7, 1, 9, 4 },
	{ 7, 9, 3, 1, 13, 12, 11, 14, 2, 6, 5, 10, 4, 0, 15, 8 },
	{ 9, 0, 5, 7, 2, 4, 10, 15, 14, 1, 11, 12, 6, 8, 3, 13 },
	{ 2, 12, 6, 10, 0, 11, 8, 3, 4, 13, 7, 5, 15, 14, 1, 9 }
};

__device__ __constant__
static const uint32_t __align__(32) c_IV256[8] = {
	SPH_C32(0x6A09E667), SPH_C32(0xBB67AE85),
	SPH_C32(0x3C6EF372), SPH_C32(0xA54FF53A),
	SPH_C32(0x510E527F), SPH_C32(0x9B05688C),
	SPH_C32(0x1F83D9AB), SPH_C32(0x5BE0CD19)
};

__device__ __constant__
static const uint32_t __align__(32) c_u256[16] = {
	SPH_C32(0x243F6A88), SPH_C32(0x85A308D3),
	SPH_C32(0x13198A2E), SPH_C32(0x03707344),
	SPH_C32(0xA4093822), SPH_C32(0x299F31D0),
	SPH_C32(0x082EFA98), SPH_C32(0xEC4E6C89),
	SPH_C32(0x452821E6), SPH_C32(0x38D01377),
	SPH_C32(0xBE5466CF), SPH_C32(0x34E90C6C),
	SPH_C32(0xC0AC29B7), SPH_C32(0xC97C50DD),
	SPH_C32(0x3F84D5B5), SPH_C32(0xB5470917)
};

#if 0
#define GS(m0, m1, c0, c1, a, b, c, d)   do { \
		a = SPH_T32(a + b + (m0 ^ c1)); \
		d = SPH_ROTR32(d ^ a, 16); \
		c = SPH_T32(c + d); \
		b = SPH_ROTR32(b ^ c, 12); \
		a = SPH_T32(a + b + (m1 ^ c0)); \
		d = SPH_ROTR32(d ^ a, 8); \
		c = SPH_T32(c + d); \
		b = SPH_ROTR32(b ^ c, 7); \
	} while (0)

#define ROUND_S(r)   do { \
	GS(Mx(r, 0x0), Mx(r, 0x1), CSx(r, 0x0), CSx(r, 0x1), v[0], v[4], v[0x8], v[0xC]); \
	GS(Mx(r, 0x2), Mx(r, 0x3), CSx(r, 0x2), CSx(r, 0x3), v[1], v[5], v[0x9], v[0xD]); \
	GS(Mx(r, 0x4), Mx(r, 0x5), CSx(r, 0x4), CSx(r, 0x5), v[2], v[6], v[0xA], v[0xE]); \
	GS(Mx(r, 0x6), Mx(r, 0x7), CSx(r, 0x6), CSx(r, 0x7), v[3], v[7], v[0xB], v[0xF]); \
	GS(Mx(r, 0x8), Mx(r, 0x9), CSx(r, 0x8), CSx(r, 0x9), v[0], v[5], v[0xA], v[0xF]); \
	GS(Mx(r, 0xA), Mx(r, 0xB), CSx(r, 0xA), CSx(r, 0xB), v[1], v[6], v[0xB], v[0xC]); \
	GS(Mx(r, 0xC), Mx(r, 0xD), CSx(r, 0xC), CSx(r, 0xD), v[2], v[7], v[0x8], v[0xD]); \
	GS(Mx(r, 0xE), Mx(r, 0xF), CSx(r, 0xE), CSx(r, 0xF), v[3], v[4], v[0x9], v[0xE]); \
} while (0)
#endif

#define GS(a,b,c,d,x) { \
	const uint32_t idx1 = c_sigma[i][x]; \
	const uint32_t idx2 = c_sigma[i][x+1]; \
	v[a] += (m[idx1] ^ c_u256[idx2]) + v[b]; \
	v[d] = SPH_ROTL32(v[d] ^ v[a], 16); \
	v[c] += v[d]; \
	v[b] = SPH_ROTR32(v[b] ^ v[c], 12); \
\
	v[a] += (m[idx2] ^ c_u256[idx1]) + v[b]; \
	v[d] = SPH_ROTR32(v[d] ^ v[a], 8); \
	v[c] += v[d]; \
	v[b] = SPH_ROTR32(v[b] ^ v[c], 7); \
}

/* Second part (64-80) msg never change, store it */
__device__ __constant__
static const uint32_t __align__(32) c_Padding[16] = {
	0, 0, 0, 0,
	0x80000000UL, 0, 0, 0,
	0, 0, 0, 0,
	0, 1, 0, 640,
};

__device__ static
void blake256_compress(uint32_t *h, const uint32_t *block, const uint32_t T0, int blakerounds)
{
	uint32_t /* __align__(8) */ m[16];
	uint32_t /* __align__(8) */ v[16];

	m[0] = block[0];
	m[1] = block[1];
	m[2] = block[2];
	m[3] = block[3];

	for (uint32_t i = 4; i < 16; i++) {
		m[i] = (T0 == 0x200) ? block[i] : c_Padding[i];
	}

	//#pragma unroll 8
	for(uint32_t i = 0; i < 8; i++)
		v[i] = h[i];

	v[ 8] = c_u256[0];
	v[ 9] = c_u256[1];
	v[10] = c_u256[2];
	v[11] = c_u256[3];

	v[12] = c_u256[4] ^ T0;
	v[13] = c_u256[5] ^ T0;
	v[14] = c_u256[6];
	v[15] = c_u256[7];

	for (int i = 0; i < blakerounds; i++) {
		/* column step */
		GS(0, 4, 0x8, 0xC, 0x0);
		GS(1, 5, 0x9, 0xD, 0x2);
		GS(2, 6, 0xA, 0xE, 0x4);
		GS(3, 7, 0xB, 0xF, 0x6);
		/* diagonal step */
		GS(0, 5, 0xA, 0xF, 0x8);
		GS(1, 6, 0xB, 0xC, 0xA);
		GS(2, 7, 0x8, 0xD, 0xC);
		GS(3, 4, 0x9, 0xE, 0xE);
	}

	//#pragma unroll 16
	for (uint32_t i = 0; i < 16; i++) {
		uint32_t j = i % 8;
		h[j] ^= v[i];
	}
}

__global__
void blake256_gpu_hash_80(uint32_t threads, uint32_t startNounce, uint32_t *resNounce, const int blakerounds, const int crcsum)
{
	uint32_t thread = (blockDim.x * blockIdx.x + threadIdx.x);
	if (thread < threads)
	{
		const uint32_t nounce = startNounce + thread;
		uint32_t h[8];

		#pragma unroll
		for(int i=0; i<8; i++) {
			h[i] = c_IV256[i];
		}

#if !USE_CACHE
		blake256_compress(h, c_data, 512, blakerounds);
#else
		if (crcsum != prevsum) {
			prevsum = crcsum;
			blake256_compress(h, c_data, 512, blakerounds);
			#pragma unroll
			for(int i=0; i<8; i++) {
				cache[i] = h[i];
			}
		} else {
			#pragma unroll
			for(int i=0; i<8; i++) {
				h[i] = cache[i];
			}
		}
#endif
		// ------ Close: Bytes 64 to 80 ------ 

		uint32_t ending[4];
		ending[0] = c_data[16];
		ending[1] = c_data[17];
		ending[2] = c_data[18];
		ending[3] = nounce; /* our tested value */

		blake256_compress(h, ending, 640, blakerounds);

		for (int i = 7; i >= 0; i--) {
			uint32_t hash = cuda_swab32(h[i]);
			if (hash > c_Target[i]) {
				return;
			}
			if (hash < c_Target[i]) {
				break;
			}
		}

		/* keep the smallest nounce, + extra one if found */
		if (resNounce[0] > nounce) {
			resNounce[1] = resNounce[0];
			resNounce[0] = nounce;
		}
		else
			resNounce[1] = nounce;
	}
}

__host__
uint32_t blake256_cpu_hash_80(int thr_id, uint32_t threads, uint32_t startNounce, const int blakerounds, const uint32_t crcsum)
{
	const int threadsperblock = TPB;
	uint32_t result = MAXU;

	dim3 grid((threads + threadsperblock-1)/threadsperblock);
	dim3 block(threadsperblock);
	size_t shared_size = 0;

	/* Check error on Ctrl+C or kill to prevent segfaults on exit */
	if (hipMemset(d_resNounce[thr_id], 0xff, 2*sizeof(uint32_t)) != hipSuccess)
		return result;

	blake256_gpu_hash_80<<<grid, block, shared_size>>>(threads, startNounce, d_resNounce[thr_id], blakerounds, crcsum);
	hipDeviceSynchronize();
	if (hipSuccess == hipMemcpy(h_resNounce[thr_id], d_resNounce[thr_id], 2*sizeof(uint32_t), hipMemcpyDeviceToHost)) {
		hipDeviceSynchronize();
		result = h_resNounce[thr_id][0];
		extra_results[0] = h_resNounce[thr_id][1];
	}
	return result;
}

__host__
void blake256_cpu_setBlock_80(uint32_t *pdata, const uint32_t *ptarget)
{
	uint32_t data[20];
	memcpy(data, pdata, 80);
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(c_data), data, sizeof(data), 0, hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(c_sigma), host_sigma, sizeof(host_sigma), 0, hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(c_Target), ptarget, 32, 0, hipMemcpyHostToDevice));
}

extern "C" int scanhash_blake256(int thr_id, uint32_t *pdata, const uint32_t *ptarget,
	uint32_t max_nonce, unsigned long *hashes_done, uint32_t blakerounds=14)
{
	const uint32_t first_nonce = pdata[19];
	static bool init[8] = { 0, 0, 0, 0, 0, 0, 0, 0 };
	uint32_t throughput = min(TPB * 4096, max_nonce - first_nonce);
	uint32_t crcsum = MAXU;
	int rc = 0;

	if (extra_results[0] != MAXU) {
		// possible extra result found in previous call
		if (first_nonce <= extra_results[0] && max_nonce >= extra_results[0]) {
			pdata[19] = extra_results[0];
			*hashes_done = pdata[19] - first_nonce + 1;
			extra_results[0] = MAXU;
			rc = 1;
			goto exit_scan;
		}
	}

	if (opt_benchmark)
		((uint32_t*)ptarget)[7] = 0x00000f;

	if (!init[thr_id]) {
		if (opt_n_threads > 1) {
			CUDA_SAFE_CALL(hipSetDevice(device_map[thr_id]));
		}
		CUDA_SAFE_CALL(hipHostMalloc(&h_resNounce[thr_id], 2*sizeof(uint32_t)));
		CUDA_SAFE_CALL(hipMalloc(&d_resNounce[thr_id], 2*sizeof(uint32_t)));
		init[thr_id] = true;
	}

	if (opt_debug && throughput < (TPB * 4096))
		applog(LOG_DEBUG, "throughput=%u, start=%x, max=%x", throughput, first_nonce, max_nonce);

	blake256_cpu_setBlock_80(pdata, ptarget);
#if USE_CACHE
	crcsum = crc32_u32t(pdata, 64);
#endif

	do {
		// GPU HASH
		uint32_t foundNonce = blake256_cpu_hash_80(thr_id, throughput, pdata[19], blakerounds, crcsum);
		if (foundNonce != MAXU)
		{
			uint32_t endiandata[20];
			uint32_t vhashcpu[8];
			uint32_t Htarg = ptarget[7];

			for (int k=0; k < 19; k++)
				be32enc(&endiandata[k], pdata[k]);

			be32enc(&endiandata[19], foundNonce);

			blake256hash(vhashcpu, endiandata, blakerounds);

			if (vhashcpu[7] <= Htarg && fulltest(vhashcpu, ptarget))
			{
				pdata[19] = foundNonce;
				rc = 1;

				if (extra_results[0] != MAXU) {
					// Rare but possible if the throughput is big
					be32enc(&endiandata[19], extra_results[0]);
					blake256hash(vhashcpu, endiandata, blakerounds);
					if (vhashcpu[7] <= Htarg && fulltest(vhashcpu, ptarget)) {
						applog(LOG_NOTICE, "GPU found more than one result yippee!");
						rc = 2;
					} else {
						extra_results[0] = MAXU;
					}
				}

				goto exit_scan;
			}
			else if (vhashcpu[7] > Htarg) {
				applog(LOG_WARNING, "GPU #%d: result for nounce %08x is not in range: %x > %x", thr_id, foundNonce, vhashcpu[7], Htarg);
			}
			else if (vhashcpu[6] > ptarget[6]) {
				applog(LOG_WARNING, "GPU #%d: hash[6] for nounce %08x is not in range: %x > %x", thr_id, foundNonce, vhashcpu[6], ptarget[6]);
			}
			else {
				applog(LOG_WARNING, "GPU #%d: result for nounce %08x does not validate on CPU!", thr_id, foundNonce);
			}
		}

		if ((uint64_t) pdata[19] + throughput > (uint64_t) max_nonce) {
			pdata[19] = max_nonce - first_nonce + 1;
			break;
		}

		pdata[19] += throughput;

	} while (!work_restart[thr_id].restart);

exit_scan:
	*hashes_done = pdata[19] - first_nonce + 1;
#if 0
	/* reset the device to allow multiple instances
	 * could be made in cpu-miner... check later if required */
	if (opt_n_threads == 1) {
		CUDA_SAFE_CALL(hipDeviceReset());
		init[thr_id] = false;
	}
#endif
	// wait proper end of all threads
	//hipDeviceSynchronize();
	return rc;
}
