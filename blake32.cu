#include "hip/hip_runtime.h"
/**
 * Blake-256 Cuda Kernel (Tested on SM 5.0)
 *
 * Tanguy Pruvot - Aug. 2014
 */

#include "miner.h"

extern "C" {
#include "sph/sph_blake.h"
#include <stdint.h>
#include <memory.h>
}

/* threads per block */
#define TPB 128

/* hash by cpu with blake 256 */
extern "C" void blake32hash(void *output, const void *input)
{
	unsigned char hash[64];
	sph_blake256_context ctx;
	sph_blake256_init(&ctx);
	sph_blake256(&ctx, input, 80);
	sph_blake256_close(&ctx, hash);
	memcpy(output, hash, 32);
}

#include "cuda_helper.h"

// in cpu-miner.c
extern bool opt_n_threads;
extern bool opt_benchmark;
//extern bool opt_debug;
extern int device_map[8];

extern hipError_t MyStreamSynchronize(hipStream_t stream, int situation, int thr_id);

__constant__
static uint32_t c_Target[8];

__constant__
static uint32_t __align__(32) c_PaddedMessage80[32]; // padded message (80 bytes + padding)

static uint32_t *d_resNounce[8];
static uint32_t *h_resNounce[8];

__constant__
static uint8_t c_sigma[16][16];
const uint8_t host_sigma[16][16] =
{
  { 0, 1, 2, 3, 4, 5, 6, 7, 8, 9, 10, 11, 12, 13, 14, 15 },
  {14, 10, 4, 8, 9, 15, 13, 6, 1, 12, 0, 2, 11, 7, 5, 3 },
  {11, 8, 12, 0, 5, 2, 15, 13, 10, 14, 3, 6, 7, 1, 9, 4 },
  { 7, 9, 3, 1, 13, 12, 11, 14, 2, 6, 5, 10, 4, 0, 15, 8 },
  { 9, 0, 5, 7, 2, 4, 10, 15, 14, 1, 11, 12, 6, 8, 3, 13 },
  { 2, 12, 6, 10, 0, 11, 8, 3, 4, 13, 7, 5, 15, 14, 1, 9 },
  {12, 5, 1, 15, 14, 13, 4, 10, 0, 7, 6, 3, 9, 2, 8, 11 },
  {13, 11, 7, 14, 12, 1, 3, 9, 5, 0, 15, 4, 8, 6, 2, 10 },
  { 6, 15, 14, 9, 11, 3, 0, 8, 12, 2, 13, 7, 1, 4, 10, 5 },
  {10, 2, 8, 4, 7, 6, 1, 5, 15, 11, 9, 14, 3, 12, 13 , 0 },
  { 0, 1, 2, 3, 4, 5, 6, 7, 8, 9, 10, 11, 12, 13, 14, 15 },
  {14, 10, 4, 8, 9, 15, 13, 6, 1, 12, 0, 2, 11, 7, 5, 3 },
  {11, 8, 12, 0, 5, 2, 15, 13, 10, 14, 3, 6, 7, 1, 9, 4 },
  { 7, 9, 3, 1, 13, 12, 11, 14, 2, 6, 5, 10, 4, 0, 15, 8 },
  { 9, 0, 5, 7, 2, 4, 10, 15, 14, 1, 11, 12, 6, 8, 3, 13 },
  { 2, 12, 6, 10, 0, 11, 8, 3, 4, 13, 7, 5, 15, 14, 1, 9 }
};

__device__ __constant__
static const uint32_t c_IV256[8] = {
	SPH_C32(0x6A09E667), SPH_C32(0xBB67AE85),
	SPH_C32(0x3C6EF372), SPH_C32(0xA54FF53A),
	SPH_C32(0x510E527F), SPH_C32(0x9B05688C),
	SPH_C32(0x1F83D9AB), SPH_C32(0x5BE0CD19)
};

__device__ __constant__

static const uint32_t c_u256[16] = {
	SPH_C32(0x243F6A88), SPH_C32(0x85A308D3),
	SPH_C32(0x13198A2E), SPH_C32(0x03707344),
	SPH_C32(0xA4093822), SPH_C32(0x299F31D0),
	SPH_C32(0x082EFA98), SPH_C32(0xEC4E6C89),
	SPH_C32(0x452821E6), SPH_C32(0x38D01377),
	SPH_C32(0xBE5466CF), SPH_C32(0x34E90C6C),
	SPH_C32(0xC0AC29B7), SPH_C32(0xC97C50DD),
	SPH_C32(0x3F84D5B5), SPH_C32(0xB5470917)
};

#if 0
#define GS(m0, m1, c0, c1, a, b, c, d)   do { \
		a = SPH_T32(a + b + (m0 ^ c1)); \
		d = SPH_ROTR32(d ^ a, 16); \
		c = SPH_T32(c + d); \
		b = SPH_ROTR32(b ^ c, 12); \
		a = SPH_T32(a + b + (m1 ^ c0)); \
		d = SPH_ROTR32(d ^ a, 8); \
		c = SPH_T32(c + d); \
		b = SPH_ROTR32(b ^ c, 7); \
	} while (0)

#define ROUND_S(r)   do { \
	GS(Mx(r, 0x0), Mx(r, 0x1), CSx(r, 0x0), CSx(r, 0x1), v[0], v[4], v[0x8], v[0xC]); \
	GS(Mx(r, 0x2), Mx(r, 0x3), CSx(r, 0x2), CSx(r, 0x3), v[1], v[5], v[0x9], v[0xD]); \
	GS(Mx(r, 0x4), Mx(r, 0x5), CSx(r, 0x4), CSx(r, 0x5), v[2], v[6], v[0xA], v[0xE]); \
	GS(Mx(r, 0x6), Mx(r, 0x7), CSx(r, 0x6), CSx(r, 0x7), v[3], v[7], v[0xB], v[0xF]); \
	GS(Mx(r, 0x8), Mx(r, 0x9), CSx(r, 0x8), CSx(r, 0x9), v[0], v[5], v[0xA], v[0xF]); \
	GS(Mx(r, 0xA), Mx(r, 0xB), CSx(r, 0xA), CSx(r, 0xB), v[1], v[6], v[0xB], v[0xC]); \
	GS(Mx(r, 0xC), Mx(r, 0xD), CSx(r, 0xC), CSx(r, 0xD), v[2], v[7], v[0x8], v[0xD]); \
	GS(Mx(r, 0xE), Mx(r, 0xF), CSx(r, 0xE), CSx(r, 0xF), v[3], v[4], v[0x9], v[0xE]); \
} while (0)
#endif

#define GS(a,b,c,d,e) { \
	v[a] += (m[sigma[i][e]] ^ u256[sigma[i][e+1]]) + v[b]; \
	v[d] = SPH_ROTR32(v[d] ^ v[a], 16); \
	v[c] += v[d]; \
	v[b] = SPH_ROTR32(v[b] ^ v[c], 12); \
\
	v[a] += (m[sigma[i][e+1]] ^ u256[sigma[i][e]]) + v[b]; \
	v[d] = SPH_ROTR32(v[d] ^ v[a], 8); \
	v[c] += v[d]; \
	v[b] = SPH_ROTR32(v[b] ^ v[c], 7); \
}

#define BLAKE256_ROUNDS 14

__device__ static
void blake256_compress(uint32_t *h, uint32_t *block, uint8_t ((*sigma)[16]), const uint32_t *u256, const uint32_t T0, uint8_t nullt = 1)
{
	uint32_t /* __align__(8) */ v[16];
	uint32_t /* __align__(8) */ m[16];

	//#pragma unroll
	for (int i = 0; i < 16; ++i) {
		m[i] = block[i];
	}

	//#pragma unroll 8
	for(int i = 0; i < 8; i++)
		v[i] = h[i];

	v[ 8] = u256[0];
	v[ 9] = u256[1];
	v[10] = u256[2];
	v[11] = u256[3];

	v[12] = u256[4] ^ T0;
	v[13] = u256[5] ^ T0;
	v[14] = u256[6];
	v[15] = u256[7];

	//#pragma unroll
	for (int i = 0; i < BLAKE256_ROUNDS; i++) {
		/* column step */
		GS(0, 4, 0x8, 0xC, 0);
		GS(1, 5, 0x9, 0xD, 2);
		GS(2, 6, 0xA, 0xE, 4);
		GS(3, 7, 0xB, 0xF, 6);
		/* diagonal step */
		GS(0, 5, 0xA, 0xF, 0x8);
		GS(1, 6, 0xB, 0xC, 0xA);
		GS(2, 7, 0x8, 0xD, 0xC);
		GS(3, 4, 0x9, 0xE, 0xE);
	}

	//#pragma unroll 16
	for(int i = 0; i < 16; i++)
		h[i % 8] ^= v[i];
}

#if __CUDA_ARCH__ >= 200
/* memory should be aligned to use __nvvm_memset */
#if (__NV_POINTER_SIZE == 64)
# define SZCT uint64_t
#else
# define SZCT uint32_t
#endif
extern __device__ __device_builtin__ void __nvvm_memset(uint8_t *, unsigned char, SZCT, int);
#endif

__global__
void blake256_gpu_hash_80(uint32_t threads, uint32_t startNounce, uint32_t *resNounce)
{
	uint32_t thread = (blockDim.x * blockIdx.x + threadIdx.x);
	if (thread < threads)
	{
		const uint32_t nounce = startNounce + thread;
		uint32_t /* __align__(8) */ msg[16];
		uint32_t h[8];

		#pragma unroll
		for(int i=0; i<8; i++)
			h[i] = c_IV256[i];

		blake256_compress(h, c_PaddedMessage80, c_sigma, c_u256, 0x200); /* 512 = 0x200 */

		// ------ Close: Bytes 64 to 80 ------ 

		msg[0] = c_PaddedMessage80[16];
		msg[1] = c_PaddedMessage80[17];
		msg[2] = c_PaddedMessage80[18];
		msg[3] = nounce; /* our tested value */
		msg[4] = 0x80000000UL; //cuda_swab32(0x80U);

		msg[5] = 0;  // uchar[17 to 55]
		msg[6] = 0;
		msg[7] = 0;
		msg[8] = 0;
		msg[9] = 0;
		msg[10] = 0;
		msg[11] = 0;
		msg[12] = 0;

		msg[13] = 1;
		msg[14] = 0;
		msg[15] = 0x280;

		blake256_compress(h, msg, c_sigma, c_u256, 0x280);

		for (int i = 7; i >= 0; i--) {
			uint32_t hash = cuda_swab32(h[i]);
			if (hash > c_Target[i]) {
				return;
			}
			if (hash < c_Target[i]) {
				break;
			}
		}

		/* keep the smallest nounce, hmm... */
		if(resNounce[0] > nounce)
			resNounce[0] = nounce;
	}
}

__host__
uint32_t blake256_cpu_hash_80(int thr_id, uint32_t threads, uint32_t startNounce)
{
	const int threadsperblock = TPB;

	dim3 grid((threads + threadsperblock-1)/threadsperblock);
	dim3 block(threadsperblock);
	size_t shared_size = 0;

	uint32_t result = 0xffffffffU;
	hipMemset(d_resNounce[thr_id], 0xff, sizeof(uint32_t));

	blake256_gpu_hash_80<<<grid, block, shared_size>>>(threads, startNounce, d_resNounce[thr_id]);
	MyStreamSynchronize(NULL, 1, thr_id);

	if (hipSuccess == hipMemcpy(h_resNounce[thr_id], d_resNounce[thr_id], sizeof(uint32_t), hipMemcpyDeviceToHost)) {
		hipDeviceSynchronize();
		result = *h_resNounce[thr_id];
	}
	return result;
}

__host__
void blake256_cpu_setBlock_80(uint32_t *pdata, const void *ptarget)
{
	uint32_t PaddedMessage[32];
	memcpy(PaddedMessage, pdata, 80);
	memset(&PaddedMessage[20], 0, 48);
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(c_PaddedMessage80), PaddedMessage, sizeof(PaddedMessage), 0, hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(c_sigma), host_sigma, sizeof(host_sigma), 0, hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(c_Target), ptarget, 32, 0, hipMemcpyHostToDevice));
}

extern "C" int scanhash_blake32(int thr_id, uint32_t *pdata, const uint32_t *ptarget,
	uint32_t max_nonce, unsigned long *hashes_done)
{
	const uint32_t first_nonce = pdata[19];
	static bool init[8] = { 0, 0, 0, 0, 0, 0, 0, 0 };
	uint32_t throughput = min(TPB * 2048, max_nonce - first_nonce);
	int rc = 0;

	if (opt_benchmark)
		((uint32_t*)ptarget)[7] = 0x00000f;

	if (!init[thr_id]) {
		if (opt_n_threads > 1) {
			CUDA_SAFE_CALL(hipSetDevice(device_map[thr_id]));
		}
		CUDA_SAFE_CALL(hipHostMalloc(&h_resNounce[thr_id], sizeof(uint32_t)));
		CUDA_SAFE_CALL(hipMalloc(&d_resNounce[thr_id], sizeof(uint32_t)));
		init[thr_id] = true;
	}

	if (throughput < (TPB * 2048))
		applog(LOG_WARNING, "throughput=%u, start=%x, max=%x", throughput, first_nonce, max_nonce);

	blake256_cpu_setBlock_80(pdata, (void*)ptarget);

	do {
		// GPU HASH
		uint32_t foundNonce = blake256_cpu_hash_80(thr_id, throughput, pdata[19]);
		if (foundNonce != 0xffffffff)
		{
			uint32_t endiandata[20];
			uint32_t vhashcpu[8];
			uint32_t Htarg = ptarget[7];

			for (int k=0; k < 20; k++)
				be32enc(&endiandata[k], pdata[k]);

			if (opt_debug && !opt_quiet) {
				applog(LOG_DEBUG, "throughput=%u, start=%x, max=%x, pdata=%08x...%08x",
					throughput, first_nonce, max_nonce, endiandata[0], endiandata[7]);
				applog_hash((unsigned char *)pdata);
			}

			be32enc(&endiandata[19], foundNonce);

			blake32hash(vhashcpu, endiandata);

			if (vhashcpu[7] <= Htarg && fulltest(vhashcpu, ptarget))
			{
				pdata[19] = foundNonce;
				rc = 1;
				goto exit_scan;
			}
			else if (vhashcpu[7] > Htarg) {
				applog(LOG_WARNING, "GPU #%d: result for nounce %08x is not in range: %x > %x", thr_id, foundNonce, vhashcpu[7], Htarg);
			}
			else if (vhashcpu[6] > ptarget[6]) {
				applog(LOG_WARNING, "GPU #%d: hash[6] for nounce %08x is not in range: %x > %x", thr_id, foundNonce, vhashcpu[6], ptarget[6]);
			}
			else {
				applog(LOG_WARNING, "GPU #%d: result for nounce %08x does not validate on CPU!", thr_id, foundNonce);
			}
		}

		pdata[19] += throughput;

	} while (pdata[19] < max_nonce && !work_restart[thr_id].restart);

exit_scan:
	*hashes_done = pdata[19] - first_nonce + 1;
	// reset the device to allow multiple instances
	if (opt_n_threads == 1) {
		CUDA_SAFE_CALL(hipDeviceReset());
		init[thr_id] = false;
	}
	// wait proper end of all threads
	hipDeviceSynchronize();
	return rc;
}
