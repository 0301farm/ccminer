#include "hip/hip_runtime.h"
/**
 * Blake-256 Cuda Kernel (Tested on SM 5.0)
 *
 * Tanguy Pruvot - Aug. 2014
 */

#include "miner.h"

extern "C" {
#include "sph/sph_blake.h"
#include <stdint.h>
#include <memory.h>
}

/* hash by cpu with blake 256 */
extern "C" void blake32hash(void *output, const void *input)
{
	unsigned char hash[64];
	sph_blake256_context ctx;
	sph_blake256_init(&ctx);
	sph_blake256(&ctx, input, 80);
	sph_blake256_close(&ctx, hash);
	memcpy(output, hash, 32);
}

#include "cuda_helper.h"

#if __CUDA_ARCH__ < 350
	// Kepler (Compute 3.0) + Host
	#define ROTR32(x, n) (((x) >> (n)) | ((x) << (32 - (n))))
#else
	// Kepler (Compute 3.5 / 5.0)
	#define ROTR32(x, n) __funnelshift_r( (x), (x), (n) )
#endif

// in cpu-miner.c
extern bool opt_benchmark;
extern bool opt_debug;
extern int device_map[8];

extern hipError_t MyStreamSynchronize(hipStream_t stream, int situation, int thr_id);

// shared for 8 threads of addresses (hipMalloc)
uint32_t* d_hash[8];

__constant__
static uint32_t pTarget[8];

__constant__
static uint32_t c_PaddedMessage80[32]; // padded message (80 bytes + padding)
static uint32_t *d_resNounce[8];
static uint32_t *h_resNounce[8];

__constant__
static uint8_t c_sigma[16][16];
const uint8_t host_sigma[16][16] =
{
  { 0, 1, 2, 3, 4, 5, 6, 7, 8, 9, 10, 11, 12, 13, 14, 15 },
  {14, 10, 4, 8, 9, 15, 13, 6, 1, 12, 0, 2, 11, 7, 5, 3 },
  {11, 8, 12, 0, 5, 2, 15, 13, 10, 14, 3, 6, 7, 1, 9, 4 },
  { 7, 9, 3, 1, 13, 12, 11, 14, 2, 6, 5, 10, 4, 0, 15, 8 },
  { 9, 0, 5, 7, 2, 4, 10, 15, 14, 1, 11, 12, 6, 8, 3, 13 },
  { 2, 12, 6, 10, 0, 11, 8, 3, 4, 13, 7, 5, 15, 14, 1, 9 },
  {12, 5, 1, 15, 14, 13, 4, 10, 0, 7, 6, 3, 9, 2, 8, 11 },
  {13, 11, 7, 14, 12, 1, 3, 9, 5, 0, 15, 4, 8, 6, 2, 10 },
  { 6, 15, 14, 9, 11, 3, 0, 8, 12, 2, 13, 7, 1, 4, 10, 5 },
  {10, 2, 8, 4, 7, 6, 1, 5, 15, 11, 9, 14, 3, 12, 13 , 0 },
  { 0, 1, 2, 3, 4, 5, 6, 7, 8, 9, 10, 11, 12, 13, 14, 15 },
  {14, 10, 4, 8, 9, 15, 13, 6, 1, 12, 0, 2, 11, 7, 5, 3 },
  {11, 8, 12, 0, 5, 2, 15, 13, 10, 14, 3, 6, 7, 1, 9, 4 },
  { 7, 9, 3, 1, 13, 12, 11, 14, 2, 6, 5, 10, 4, 0, 15, 8 },
  { 9, 0, 5, 7, 2, 4, 10, 15, 14, 1, 11, 12, 6, 8, 3, 13 },
  { 2, 12, 6, 10, 0, 11, 8, 3, 4, 13, 7, 5, 15, 14, 1, 9 }
};

__device__ __constant__
static const uint32_t c_IV256[8] = {
	SPH_C32(0x6A09E667), SPH_C32(0xBB67AE85),
	SPH_C32(0x3C6EF372), SPH_C32(0xA54FF53A),
	SPH_C32(0x510E527F), SPH_C32(0x9B05688C),
	SPH_C32(0x1F83D9AB), SPH_C32(0x5BE0CD19)
};

__device__ __constant__

static const uint32_t c_u256[16] = {
	SPH_C32(0x243F6A88), SPH_C32(0x85A308D3),
	SPH_C32(0x13198A2E), SPH_C32(0x03707344),
	SPH_C32(0xA4093822), SPH_C32(0x299F31D0),
	SPH_C32(0x082EFA98), SPH_C32(0xEC4E6C89),
	SPH_C32(0x452821E6), SPH_C32(0x38D01377),
	SPH_C32(0xBE5466CF), SPH_C32(0x34E90C6C),
	SPH_C32(0xC0AC29B7), SPH_C32(0xC97C50DD),
	SPH_C32(0x3F84D5B5), SPH_C32(0xB5470917)
};

#if 0
#define GS(m0, m1, c0, c1, a, b, c, d)   do { \
		a = SPH_T32(a + b + (m0 ^ c1)); \
		d = SPH_ROTR32(d ^ a, 16); \
		c = SPH_T32(c + d); \
		b = SPH_ROTR32(b ^ c, 12); \
		a = SPH_T32(a + b + (m1 ^ c0)); \
		d = SPH_ROTR32(d ^ a, 8); \
		c = SPH_T32(c + d); \
		b = SPH_ROTR32(b ^ c, 7); \
	} while (0)

#define ROUND_S(r)   do { \
	GS(Mx(r, 0x0), Mx(r, 0x1), CSx(r, 0x0), CSx(r, 0x1), v[0], v[4], v[0x8], v[0xC]); \
	GS(Mx(r, 0x2), Mx(r, 0x3), CSx(r, 0x2), CSx(r, 0x3), v[1], v[5], v[0x9], v[0xD]); \
	GS(Mx(r, 0x4), Mx(r, 0x5), CSx(r, 0x4), CSx(r, 0x5), v[2], v[6], v[0xA], v[0xE]); \
	GS(Mx(r, 0x6), Mx(r, 0x7), CSx(r, 0x6), CSx(r, 0x7), v[3], v[7], v[0xB], v[0xF]); \
	GS(Mx(r, 0x8), Mx(r, 0x9), CSx(r, 0x8), CSx(r, 0x9), v[0], v[5], v[0xA], v[0xF]); \
	GS(Mx(r, 0xA), Mx(r, 0xB), CSx(r, 0xA), CSx(r, 0xB), v[1], v[6], v[0xB], v[0xC]); \
	GS(Mx(r, 0xC), Mx(r, 0xD), CSx(r, 0xC), CSx(r, 0xD), v[2], v[7], v[0x8], v[0xD]); \
	GS(Mx(r, 0xE), Mx(r, 0xF), CSx(r, 0xE), CSx(r, 0xF), v[3], v[4], v[0x9], v[0xE]); \
} while (0)
#endif

#define GS(a,b,c,d,e) { \
	v[a] += (m[sigma[i][e]] ^ u256[sigma[i][e+1]]) + v[b]; \
	v[d] = ROTR32(v[d] ^ v[a], 16); \
	v[c] += v[d]; \
	v[b] = ROTR32(v[b] ^ v[c], 12); \
\
	v[a] += (m[sigma[i][e+1]] ^ u256[sigma[i][e]]) + v[b]; \
	v[d] = ROTR32(v[d] ^ v[a], 8); \
	v[c] += v[d]; \
	v[b] = ROTR32(v[b] ^ v[c], 7); \
}

__device__ static
void blake256_compress(uint32_t *h, uint32_t *block, uint8_t ((*sigma)[16]), const uint32_t *u256, const uint32_t T0, uint8_t nullt = 1)
{
	uint32_t /* __align__(8) */ v[16];
	uint32_t /* __align__(8) */ m[16];

	//#pragma unroll
	for (int i = 0; i < 16; ++i) {
		m[i] = cuda_swab32(block[i]);
		//m[i] = block[i];
	}

	#pragma unroll
	for(int i = 0; i < 8; i++)
		v[i] = h[i];

	v[ 8] = u256[0];
	v[ 9] = u256[1];
	v[10] = u256[2];
	v[11] = u256[3];

	v[12] = u256[4] ^ T0;
	v[13] = u256[5] ^ T0;
	v[14] = u256[6];
	v[15] = u256[7];

	// on a 80-bytes null buffer :
	// first : v = {0x6a09e667, 0xbb67ae85, 0x3c6ef372, 0xa54ff53a, ...}
	// second : v = {0xb5bfb2f9, 0x14cfcc63, 0xb85c549c, 0xc9b4184e, ..., 0x299f3350, 0x082efa98, 0xec4e6c89}

	//#pragma unroll
	for (int i = 0; i < 14; i++) {
		/* column step */
		GS(0, 4, 0x8, 0xC, 0);
		GS(1, 5, 0x9, 0xD, 2);
		GS(2, 6, 0xA, 0xE, 4);
		GS(3, 7, 0xB, 0xF, 6);
		/* diagonal step */
		GS(0, 5, 0xA, 0xF, 0x8);
		GS(1, 6, 0xB, 0xC, 0xA);
		GS(2, 7, 0x8, 0xD, 0xC);
		GS(3, 4, 0x9, 0xE, 0xE);
	}

	//#pragma unroll 16
	for(int i = 0; i < 16; i++)
		h[i % 8] ^= v[i];

	//second H0 = 0x0c7b1594 ... H7 = 0x9051b305
}

#if __CUDA_ARCH__ >= 200
#if (__NV_POINTER_SIZE == 64)
# define SZCT uint64_t
#else
# define SZCT uint32_t
#endif
extern __device__ __device_builtin__ void __nvvm_memset(uint8_t *, unsigned char, SZCT, int);
#endif

__global__
void blake256_gpu_hash_80(int threads, uint32_t startNounce, void *outputHash)
{
	int thread = (blockDim.x * blockIdx.x + threadIdx.x);
	if (thread < threads)
	{
		uint32_t /* __align__(16) */ h[8];
		uint32_t /* __align__(16) */ msg[16];
		const uint32_t nounce = startNounce + thread;

		#pragma unroll
		for(int i=0; i<8; i++)
			h[i] = c_IV256[i];

		blake256_compress(h, c_PaddedMessage80, c_sigma, c_u256, 0x200); /* 512 = 0x200 */

		// ------ Close: Bytes 64 to 80 ------ 

#if 0 /* __CUDA_ARCH__ >= 200 */
		__nvvm_memset((uint8_t*)(&msg[4]), 0, sizeof(msg)-16, 16);
#else
		msg[5] = 0;
		msg[6] = 0;
		msg[7] = 0;
		msg[8] = 0;
		msg[9] = 0;
		msg[10] = 0;
		msg[11] = 0;
		msg[12] = 0;
		msg[14] = 0;
#endif
		msg[0] = c_PaddedMessage80[16];
		msg[1] = c_PaddedMessage80[17];
		msg[2] = c_PaddedMessage80[18];
		msg[3] = cuda_swab32(nounce); // here or at 80 ?

		msg[4] = 0x80; // uchar[16] after buffer
		msg[13] = 0x01000000; //((uint8_t*)msg)[55] = 1; // uchar[17 to 55]
		msg[15] = 0x80020000; // 60-63 0x280

		//h => {0xb5bfb2f9, 0x14cfcc63, 0xb85c549c, 0xc9b4184e, 0x67dfc6ce, 0x29e9904b, 0xd59ee74e, 0xfaa9c653}
		//msg  {0, 0, 0, 0, 0x80, 0...}

		blake256_compress(h, msg, c_sigma, c_u256, 0x280); // or 0x80
		//h => {0x0c7b1594, 0x52328517, 0x463db487, 0xdf5e39b7, 0x1322afaf, 0x14ed562c, 0xe9d18d7d, 0x9051b305}

		uint32_t *outHash = (uint32_t*) outputHash + 16*thread; // 16 = 4 x sizeof(uint32)
		//#pragma unroll
		for (int i=0; i < 8; i++) {
			outHash[i] = cuda_swab32(h[i]);
		}
	}
}

__host__
void blake256_cpu_hash_80(int thr_id, int threads, uint32_t startNounce, uint32_t *d_outputHash, int order)
{
	const int threadsperblock = 256;

	dim3 grid((threads + threadsperblock-1)/threadsperblock);
	dim3 block(threadsperblock);

	size_t shared_size = 0;

	blake256_gpu_hash_80<<<grid, block, shared_size>>>(threads, startNounce, d_outputHash);

	MyStreamSynchronize(NULL, order, thr_id);
}

__global__
void gpu_check_hash_64(int threads, uint32_t startNounce, uint32_t *g_nonceVector, uint32_t *g_hash, uint32_t *resNounce)
{
	int thread = (blockDim.x * blockIdx.x + threadIdx.x);
	if (thread < threads)
	{
		uint32_t nounce = (g_nonceVector != NULL) ? g_nonceVector[thread] : (startNounce + thread);

		int hashPosition = nounce - startNounce;
		uint32_t *inpHash = &g_hash[16 * hashPosition];
		uint32_t hash[8];

		#pragma unroll 8
		for (int i=0; i < 8; i++)
			hash[i] = inpHash[i];

		int i, position = -1;
		bool rc = true;

		#pragma unroll 8
		for (i = 7; i >= 0; i--) {
			if (hash[i] > pTarget[i] && position < i) {
				position = i;
				rc = false;
			}
			if (hash[i] < pTarget[i] && position < i) {
				position = i;
				rc = true;
			}
		}

		if(rc && resNounce[0] > nounce)
			resNounce[0] = nounce;
	}
}

__host__
uint32_t cpu_check_hash_64(int thr_id, int threads, uint32_t startNounce, uint32_t *d_nonceVector, uint32_t *d_inputHash, int order)
{
	uint32_t result = 0xffffffff;
	const int threadsperblock = 256;

	hipMemset(d_resNounce[thr_id], 0xff, sizeof(uint32_t));

	dim3 grid((threads + threadsperblock-1)/threadsperblock);
	dim3 block(threadsperblock);

	size_t shared_size = 0;

	gpu_check_hash_64 <<<grid, block, shared_size>>>(threads, startNounce, d_nonceVector, d_inputHash, d_resNounce[thr_id]);

	MyStreamSynchronize(NULL, order, thr_id);

	CUDA_SAFE_CALL(hipMemcpy(h_resNounce[thr_id], d_resNounce[thr_id], sizeof(uint32_t), hipMemcpyDeviceToHost));

	// hipMemcpy() is asynch!
	hipDeviceSynchronize();
	result = *h_resNounce[thr_id];

	return result;
}

__host__
void blake256_cpu_init(int thr_id)
{
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(c_sigma), host_sigma, sizeof(host_sigma), 0, hipMemcpyHostToDevice));

	CUDA_SAFE_CALL(hipHostMalloc(&h_resNounce[thr_id], 1*sizeof(uint32_t)));
	CUDA_SAFE_CALL(hipMalloc(&d_resNounce[thr_id], 1*sizeof(uint32_t)));
}

__host__
void blake256_cpu_setBlock_80(uint32_t *pdata, const void *ptarget)
{
	uint32_t PaddedMessage[32];
	memcpy(PaddedMessage, pdata, 80);
	memset(&PaddedMessage[20], 0, 48);
	//for (int i=0; i<20; i++)
	//	PaddedMessage[i] = cuda_swab32(pdata[i]);

	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(pTarget), ptarget, 32, 0, hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(c_PaddedMessage80), PaddedMessage, sizeof(PaddedMessage), 0, hipMemcpyHostToDevice));
}

#define NULLTEST 0

extern "C" int scanhash_blake32(int thr_id, uint32_t *pdata, const uint32_t *ptarget,
	uint32_t max_nonce, unsigned long *hashes_done)
{
	uint32_t endiandata[20];
	const uint32_t first_nonce = pdata[19];
	const int throughput = 256*256*2;
	static bool init[8] = {0,0,0,0,0,0,0,0};

	if (opt_benchmark)
		((uint32_t*)ptarget)[7] = 0x00000f;

	uint32_t Htarg = ptarget[7];

	if (!init[thr_id]) {
		CUDA_SAFE_CALL(hipSetDevice(device_map[thr_id]));
		CUDA_SAFE_CALL(hipMalloc(&d_hash[thr_id], 16 * sizeof(uint32_t) * throughput));

		blake256_cpu_init(thr_id);

		init[thr_id] = true;
	}

#if NULLTEST
	// dev test with a null buffer 0x00000...
	for (int k = 0; k < 20; k++)
		pdata[k] = 0;
	uint32_t vhash[8];
	blake32hash(vhash, pdata);
#endif

	for (int k=0; k < 20; k++)
		be32enc(&endiandata[k], pdata[k]);

	blake256_cpu_setBlock_80(endiandata, (void*)ptarget);

	do {
		int order = 0;
		uint32_t foundNonce;

		// GPU
		blake256_cpu_hash_80(thr_id, throughput, pdata[19], d_hash[thr_id], order++);

#if NULLTEST
		uint32_t buf[8]; memset(buf, 0, sizeof buf);
		CUDA_SAFE_CALL(hipMemcpy(buf, d_hash[thr_id], sizeof buf, hipMemcpyDeviceToHost));
		CUDA_SAFE_CALL(hipDeviceSynchronize());
		//applog_hash((unsigned char*)buf);
#endif
		foundNonce = cpu_check_hash_64(thr_id, throughput, pdata[19], NULL, d_hash[thr_id], order++);
		if (foundNonce != 0xffffffff)
		{
			uint32_t vhashcpu[8];
			be32enc(&endiandata[19], foundNonce);

			blake32hash(vhashcpu, endiandata);

			if (opt_debug)
				applog(LOG_DEBUG, "foundNonce = %08x",foundNonce);

			if (vhashcpu[7] <= Htarg && fulltest(vhashcpu, ptarget))
			{
				pdata[19] = foundNonce;
				*hashes_done = pdata[19] - first_nonce + 1;
				return 1;
			} else {
				applog(LOG_INFO, "GPU #%d: result for nonce %08x does not validate on CPU!", thr_id, foundNonce);
			}
		}

		pdata[19] += throughput;

	} while (pdata[19] < max_nonce && !work_restart[thr_id].restart);

	*hashes_done = pdata[19] - first_nonce + 1;
	return 0;
}

//#define DEBUG_ALGO

__host__
int scanhash_blake256_cpu(int thr_id, uint32_t *pdata, const uint32_t *ptarget,
	uint32_t max_nonce, uint64_t *hashes_done)
{
	uint32_t n = pdata[19] - 1;
	const uint32_t first_nonce = pdata[19];
	const uint32_t Htarg = ptarget[7];

	uint32_t __align__(32) hash64[8];
	uint32_t endiandata[32];

	uint64_t htmax[] = {
		0,
		0xF,
		0xFF,
		0xFFF,
		0xFFFF,
		0x10000000
	};
	uint32_t masks[] = {
		0xFFFFFFFF,
		0xFFFFFFF0,
		0xFFFFFF00,
		0xFFFFF000,
		0xFFFF0000,
		0
	};

	// we need bigendian data...
	for (int kk=0; kk < 32; kk++) {
		be32enc(&endiandata[kk], ((uint32_t*)pdata)[kk]);
	};
#ifdef DEBUG_ALGO
	if (Htarg != 0)
		printf("[%d] Htarg=%X\n", thr_id, Htarg);
#endif
	for (int m=0; m < 6; m++) {
		if (Htarg <= htmax[m]) {
			uint32_t mask = masks[m];
			do {
				pdata[19] = ++n;
				be32enc(&endiandata[19], n);
				blake32hash(hash64, endiandata);
#ifndef DEBUG_ALGO
				if ((!(hash64[7] & mask)) && fulltest(hash64, ptarget)) {
					*hashes_done = n - first_nonce + 1;
					return true;
				}
#else
				if (!(n % 0x1000) && !thr_id) printf(".");
				if (!(hash64[7] & mask)) {
					printf("[%d]",thr_id);
					if (fulltest(hash64, ptarget)) {
						*hashes_done = n - first_nonce + 1;
						return true;
					}
				}
#endif
			} while (n < max_nonce && !work_restart[thr_id].restart);
			// see blake.c if else to understand the loop on htmax => mask
			break;
		}
	}

	*hashes_done = n - first_nonce + 1;
	pdata[19] = n;
	return 0;
}
