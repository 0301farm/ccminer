#include "hip/hip_runtime.h"
/* Based on djm code */

#include <stdint.h>

#include "miner.h"
#include "cuda_helper.h"

#include <openssl/sha.h>

static uint32_t *d_hash[MAX_GPUS] ;

extern void pluck_setBlockTarget(const void* data, const void *ptarget);
extern void pluck_cpu_init(int thr_id, uint32_t threads, uint32_t *d_outputHash);
extern uint32_t pluck_cpu_hash(int thr_id, uint32_t threads, uint32_t startNounce, int order);

extern float tp_coef[MAX_GPUS];

#define ROTL(a, b) (((a) << (b)) | ((a) >> (32 - (b))))
//note, this is 64 bytes
static inline void xor_salsa8(uint32_t B[16], const uint32_t Bx[16])
{
#define ROTL(a, b) (((a) << (b)) | ((a) >> (32 - (b))))
	uint32_t x00, x01, x02, x03, x04, x05, x06, x07, x08, x09, x10, x11, x12, x13, x14, x15;
	int i;

	x00 = (B[0] ^= Bx[0]);
	x01 = (B[1] ^= Bx[1]);
	x02 = (B[2] ^= Bx[2]);
	x03 = (B[3] ^= Bx[3]);
	x04 = (B[4] ^= Bx[4]);
	x05 = (B[5] ^= Bx[5]);
	x06 = (B[6] ^= Bx[6]);
	x07 = (B[7] ^= Bx[7]);
	x08 = (B[8] ^= Bx[8]);
	x09 = (B[9] ^= Bx[9]);
	x10 = (B[10] ^= Bx[10]);
	x11 = (B[11] ^= Bx[11]);
	x12 = (B[12] ^= Bx[12]);
	x13 = (B[13] ^= Bx[13]);
	x14 = (B[14] ^= Bx[14]);
	x15 = (B[15] ^= Bx[15]);
	for (i = 0; i < 8; i += 2) {
		/* Operate on columns. */
		x04 ^= ROTL(x00 + x12, 7);  x09 ^= ROTL(x05 + x01, 7);
		x14 ^= ROTL(x10 + x06, 7);  x03 ^= ROTL(x15 + x11, 7);

		x08 ^= ROTL(x04 + x00, 9);  x13 ^= ROTL(x09 + x05, 9);
		x02 ^= ROTL(x14 + x10, 9);  x07 ^= ROTL(x03 + x15, 9);

		x12 ^= ROTL(x08 + x04, 13);  x01 ^= ROTL(x13 + x09, 13);
		x06 ^= ROTL(x02 + x14, 13);  x11 ^= ROTL(x07 + x03, 13);

		x00 ^= ROTL(x12 + x08, 18);  x05 ^= ROTL(x01 + x13, 18);
		x10 ^= ROTL(x06 + x02, 18);  x15 ^= ROTL(x11 + x07, 18);

		/* Operate on rows. */
		x01 ^= ROTL(x00 + x03, 7);  x06 ^= ROTL(x05 + x04, 7);
		x11 ^= ROTL(x10 + x09, 7);  x12 ^= ROTL(x15 + x14, 7);

		x02 ^= ROTL(x01 + x00, 9);  x07 ^= ROTL(x06 + x05, 9);
		x08 ^= ROTL(x11 + x10, 9);  x13 ^= ROTL(x12 + x15, 9);

		x03 ^= ROTL(x02 + x01, 13);  x04 ^= ROTL(x07 + x06, 13);
		x09 ^= ROTL(x08 + x11, 13);  x14 ^= ROTL(x13 + x12, 13);

		x00 ^= ROTL(x03 + x02, 18);  x05 ^= ROTL(x04 + x07, 18);
		x10 ^= ROTL(x09 + x08, 18);  x15 ^= ROTL(x14 + x13, 18);
	}
	B[0] += x00;
	B[1] += x01;
	B[2] += x02;
	B[3] += x03;
	B[4] += x04;
	B[5] += x05;
	B[6] += x06;
	B[7] += x07;
	B[8] += x08;
	B[9] += x09;
	B[10] += x10;
	B[11] += x11;
	B[12] += x12;
	B[13] += x13;
	B[14] += x14;
	B[15] += x15;
#undef ROTL
}

static void sha256_hash(uchar *hash, const uchar *data, int len)
{
	SHA256_CTX ctx;
	SHA256_Init(&ctx);
	SHA256_Update(&ctx, data, len);
	SHA256_Final(hash, &ctx);
}

// hash exactly 64 bytes (ie, sha256 block size)
static void sha256_hash512(uint32_t *hash, const uint32_t *data)
{
	uint32_t _ALIGN(64) S[16];
	uint32_t _ALIGN(64) T[16];
	uchar _ALIGN(64) E[64] = { 0 };
	int i;

	sha256_init(S);

	for (i = 0; i < 16; i++)
		T[i] = be32dec(&data[i]);
	sha256_transform(S, T, 0);

	E[3] = 0x80;
	E[61] = 0x02; // T[15] = 8 * 64 => 0x200;
	sha256_transform(S, (uint32_t*)E, 0);

	for (i = 0; i < 8; i++)
		be32enc(&hash[i], S[i]);
}

#define BLOCK_HEADER_SIZE 80
void pluckhash(uint32_t *hash, const uint32_t *data, uchar *hashbuffer, const int N)
{
	int size = N * 1024;
	sha256_hash(hashbuffer, (uchar*)data, BLOCK_HEADER_SIZE);
	memset(&hashbuffer[32], 0, 32);

	for (int i = 64; i < size - 32; i += 32)
	{
		uint32_t _ALIGN(64) randseed[16];
		uint32_t _ALIGN(64) randbuffer[16];
		uint32_t _ALIGN(64) joint[16];
		//i-4 because we use integers for all references against this, and we don't want to go 3 bytes over the defined area
		//we could use size here, but then it's probable to use 0 as the value in most cases
		int randmax = i - 4;

		//setup randbuffer to be an array of random indexes
		memcpy(randseed, &hashbuffer[i - 64], 64);

		if (i > 128) memcpy(randbuffer, &hashbuffer[i - 128], 64);
		else memset(randbuffer, 0, 64);

		xor_salsa8((uint32_t*)randbuffer, (uint32_t*)randseed);
		memcpy(joint, &hashbuffer[i - 32], 32);

		//use the last hash value as the seed
		for (int j = 32; j < 64; j += 4)
		{
			//every other time, change to next random index
			//randmax - 32 as otherwise we go beyond memory that's already been written to
			uint32_t rand = randbuffer[(j - 32) >> 2] % (randmax - 32);
			joint[j >> 2] = *((uint32_t *)&hashbuffer[rand]);
		}

		sha256_hash512((uint32_t*)&hashbuffer[i], joint);

		//setup randbuffer to be an array of random indexes
		//use last hash value and previous hash value(post-mixing)
		memcpy(randseed, &hashbuffer[i - 32], 64);

		if (i > 128) memcpy(randbuffer, &hashbuffer[i - 128], 64);
		else memset(randbuffer, 0, 64);

		xor_salsa8((uint32_t*)randbuffer, (uint32_t*)randseed);

		//use the last hash value as the seed
		for (int j = 0; j < 32; j += 2)
		{
			uint32_t rand = randbuffer[j >> 1] % randmax;
			*((uint32_t *)(hashbuffer + rand)) = *((uint32_t *)(hashbuffer + j + randmax));
		}
	}

	memcpy(hash, hashbuffer, 32);
}

static bool init[MAX_GPUS] = { 0 };

static __thread uchar* scratchbuf = NULL;

extern "C" int scanhash_pluck(int thr_id, uint32_t *pdata, const uint32_t *ptarget,
	uint32_t max_nonce, unsigned long *hashes_done)
{
	const uint32_t first_nonce = pdata[19];
	uint32_t endiandata[20];
	int opt_pluck_n = 128;

	int intensity = is_windows() ? 17 : 19; /* beware > 20 could work and create diff problems later */
	uint32_t throughput = device_intensity(thr_id, __func__, 1U << intensity);
	// divide by 128 for this algo which require a lot of memory
	throughput = throughput / 128 - 256;
	throughput = min(throughput, max_nonce - first_nonce + 1);

	if (opt_benchmark)
		((uint32_t*)ptarget)[7] = 0x0000ff;

	if (!init[thr_id])
	{
		hipSetDevice(device_map[thr_id]);
		//hipSetDeviceFlags(hipDeviceScheduleBlockingSync);
		//hipDeviceSetCacheConfig(hipFuncCachePreferL1);
		hipMalloc(&d_hash[thr_id], opt_pluck_n * 1024 * throughput);

		if (!scratchbuf)
			scratchbuf = (uchar*) calloc(opt_pluck_n, 1024);

		pluck_cpu_init(thr_id, throughput, d_hash[thr_id]);

		CUDA_SAFE_CALL(hipGetLastError());
		applog(LOG_INFO, "Using %d cuda threads", throughput);

		init[thr_id] = true;
	}

	for (int k = 0; k < 20; k++)
		be32enc(&endiandata[k], ((uint32_t*)pdata)[k]);

	pluck_setBlockTarget(endiandata,ptarget);

	do {
		uint32_t foundNonce = pluck_cpu_hash(thr_id, throughput, pdata[19], 0);
		if (foundNonce != UINT32_MAX)
		{
			const uint32_t Htarg = ptarget[7];
			uint32_t vhash64[8];
			be32enc(&endiandata[19], foundNonce);
			pluckhash(vhash64, endiandata, scratchbuf, opt_pluck_n);
			if (vhash64[7] <= Htarg && fulltest(vhash64, ptarget)) {
				*hashes_done = pdata[19] - first_nonce + throughput;
				pdata[19] = foundNonce;
				return 1;
			} else {
				applog(LOG_WARNING, "GPU #%d: result for %08x does not validate on CPU!", device_map[thr_id], foundNonce);
			}
		}

		pdata[19] += throughput;

	} while (pdata[19] < max_nonce && !work_restart[thr_id].restart);

	*hashes_done = pdata[19] - first_nonce;
	return 0;
}
